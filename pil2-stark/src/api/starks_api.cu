#include "zkglobals.hpp"
#include "proof2zkinStark.hpp"
#include "starks.hpp"
#include "omp.h"
#include "starks_api.hpp"
#ifdef __USE_CUDA__
#include "gen_recursive_proof.cuh"
#include "gen_proof.cuh"
#include "gen_commit.cuh"

struct MaxSizes
{
    uint64_t maxTraceArea;
    uint64_t maxConstArea;
    uint64_t maxAuxTraceArea;
    uint64_t maxConstTreeSize;
    bool recursive;
};


void *gen_device_commit_buffers(void *maxSizes_)
{
    MaxSizes *maxSizes = (MaxSizes *)maxSizes_;
    DeviceCommitBuffers *buffers = new DeviceCommitBuffers();
    buffers->recursive = maxSizes->recursive;
    CHECKCUDAERR(hipMalloc(&buffers->d_aux_trace, maxSizes->maxAuxTraceArea * sizeof(Goldilocks::Element)));
    if(buffers->recursive) {
        CHECKCUDAERR(hipMalloc(&buffers->d_trace, maxSizes->maxTraceArea * sizeof(Goldilocks::Element)));
        CHECKCUDAERR(hipMalloc(&buffers->d_constPols, maxSizes->maxConstArea * sizeof(Goldilocks::Element)));
        CHECKCUDAERR(hipMalloc(&buffers->d_constTree, maxSizes->maxConstTreeSize * sizeof(Goldilocks::Element)));
    }
    return (void *)buffers;
}

void gen_device_commit_buffers_free(void *d_buffers)
{
    DeviceCommitBuffers *buffers = (DeviceCommitBuffers *)d_buffers;
    CHECKCUDAERR(hipFree(buffers->d_aux_trace));
    if(buffers->recursive) {
        CHECKCUDAERR(hipFree(buffers->d_trace));
        CHECKCUDAERR(hipFree(buffers->d_constPols));
        CHECKCUDAERR(hipFree(buffers->d_constTree));
    }
    delete buffers;
}

void gen_proof(void *pSetupCtx_, uint64_t airgroupId, uint64_t airId, uint64_t instanceId, void *params_, void *globalChallenge, uint64_t* proofBuffer, char *proofFile, void *d_buffers_, bool loadConstants) {

    double time = omp_get_wtime();
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    SetupCtx *setupCtx = (SetupCtx *)pSetupCtx_;
    StepsParams *params = (StepsParams *)params_;

    uint64_t N = (1 << setupCtx->starkInfo.starkStruct.nBits);
    uint64_t sizeTrace = N * (setupCtx->starkInfo.mapSectionsN["cm1"]) * sizeof(Goldilocks::Element);
    uint64_t sizeConstPols = N * (setupCtx->starkInfo.nConstants) * sizeof(Goldilocks::Element);
    uint64_t sizeConstTree = get_const_tree_size((void *)&setupCtx->starkInfo) * sizeof(Goldilocks::Element);

    uint64_t offsetStage1 = setupCtx->starkInfo.mapOffsets[std::make_pair("cm1", false)];
    uint64_t offsetConstTree = setupCtx->starkInfo.mapOffsets[std::make_pair("const", true)];
    uint64_t offsetConstPols = setupCtx->starkInfo.mapOffsets[std::make_pair("const", false)];
    CHECKCUDAERR(hipMemcpy(d_buffers->d_aux_trace + offsetStage1, params->trace, sizeTrace, hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(d_buffers->d_aux_trace + offsetConstPols, &params->aux_trace[offsetConstPols], sizeConstPols, hipMemcpyHostToDevice));
    if(loadConstants) {
        CHECKCUDAERR(hipMemcpy(d_buffers->d_aux_trace + offsetConstTree, &params->aux_trace[offsetConstTree], sizeConstTree, hipMemcpyHostToDevice));
    }

    time = omp_get_wtime() - time;
    //std::cout << "rick genDeviceBuffers time: " << time << std::endl;

    time = omp_get_wtime();
    genProof_gpu(*setupCtx, airgroupId, airId, instanceId, *params, (Goldilocks::Element *)globalChallenge, proofBuffer, string(proofFile), d_buffers);
    time = omp_get_wtime() - time;
    //std::cout << "rick genRecursiveProof_gpu time: " << time << std::endl;
}

void gen_recursive_proof(void *pSetupCtx_, char *globalInfoFile, uint64_t airgroupId, uint64_t airId, uint64_t instanceId, void *trace, void *aux_trace, void *pConstPols, void *pConstTree, void *pPublicInputs, uint64_t* proofBuffer, char *proof_file, bool vadcop, void *d_buffers_, bool loadConstants)
{

    json globalInfo;
    file2json(globalInfoFile, globalInfo);

    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    SetupCtx *setupCtx = (SetupCtx *)pSetupCtx_;
    double time = omp_get_wtime();

    uint64_t N = (1 << setupCtx->starkInfo.starkStruct.nBits);
    uint64_t sizeTrace = N * (setupCtx->starkInfo.mapSectionsN["cm1"]) * sizeof(Goldilocks::Element);
    uint64_t sizeConstPols = N * (setupCtx->starkInfo.nConstants) * sizeof(Goldilocks::Element);
    uint64_t sizeConstTree = get_const_tree_size((void *)&setupCtx->starkInfo) * sizeof(Goldilocks::Element);

    CHECKCUDAERR(hipMemcpy(d_buffers->d_trace, trace, sizeTrace, hipMemcpyHostToDevice));
    if(loadConstants) {
        CHECKCUDAERR(hipMemcpy(d_buffers->d_constPols, pConstPols, sizeConstPols, hipMemcpyHostToDevice));
        CHECKCUDAERR(hipMemcpy(d_buffers->d_constTree, pConstTree, sizeConstTree, hipMemcpyHostToDevice));
    }
    
    time = omp_get_wtime() - time;
    // std::cout << "rick genDeviceBuffers time: " << time << std::endl;

    time = omp_get_wtime();
    genRecursiveProof_gpu<Goldilocks::Element>(*setupCtx, globalInfo, airgroupId, airId, instanceId, (Goldilocks::Element *)trace, (Goldilocks::Element *)pConstPols, (Goldilocks::Element *)pConstTree, (Goldilocks::Element *)pPublicInputs, proofBuffer, string(proof_file), d_buffers, vadcop);
    time = omp_get_wtime() - time;
    // std::cout << "rick genRecursiveProof_gpu time: " << time << std::endl;
}

void commit_witness(uint64_t arity, uint64_t nBits, uint64_t nBitsExt, uint64_t nCols, void *root, void *trace, void *auxTrace, void *d_buffers_) {

    double time = omp_get_wtime();

    Goldilocks::Element *rootGL = (Goldilocks::Element *)root;
    uint64_t N = 1 << nBits;
    uint64_t NExtended = 1 << nBitsExt;


    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    uint64_t sizeTrace = N * nCols * sizeof(Goldilocks::Element);
    uint64_t offsetStage1 = 0;

    CHECKCUDAERR(hipMemcpy(d_buffers->d_aux_trace + offsetStage1, trace, sizeTrace, hipMemcpyHostToDevice));
    genCommit_gpu(arity, rootGL, N, NExtended, nCols, d_buffers);
    time = omp_get_wtime() - time;
    //std::cout << "rick genRCommit_gpu time: " << time << std::endl;
}
#endif