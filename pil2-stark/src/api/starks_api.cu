#include "zkglobals.hpp"
#include "proof2zkinStark.hpp"
#include "starks.hpp"
#include "omp.h"
#include "starks_api.hpp"
#include "starks_api_internal.hpp"
#include <cstring>
#include <thread>

#ifdef __USE_CUDA__
#include "gen_recursive_proof.cuh"
#include "gen_proof.cuh"
#include "gen_commit.cuh"
#include "poseidon2_goldilocks.cu"
#include <hip/hip_runtime.h>
#include <mutex>


struct MaxSizes
{
    uint64_t totalConstPols;
    uint64_t maxAuxTraceArea;
    uint64_t totalConstPolsAggregation;
};

uint32_t selectStream(DeviceCommitBuffers* d_buffers);
void reserveStream(DeviceCommitBuffers* d_buffers, uint32_t streamId);

void closeStreamTimer(TimerGPU &timer, bool isProve);
void get_proof(DeviceCommitBuffers *d_buffers, uint64_t streamId);
void get_commit_root(DeviceCommitBuffers *d_buffers, uint64_t streamId);



void *gen_device_buffers(void *maxSizes_, uint32_t node_rank, uint32_t node_size)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount % node_size != 0) {
        zklog.error("Device count must be divisible by numer of processe per node");
        exit(1);
    }
    if (deviceCount < node_size) {
        zklog.error("Numer of processes per node can not be greater than number of GPUs");
        exit(1);
    }
    MaxSizes *maxSizes = (MaxSizes *)maxSizes_;

    DeviceCommitBuffers *d_buffers = new DeviceCommitBuffers();
    d_buffers->n_gpus = (uint32_t) deviceCount / node_size;
    d_buffers->gpus_g2l = (uint32_t *)malloc(deviceCount * sizeof(uint32_t));
    d_buffers->my_gpu_ids = (uint32_t *)malloc(d_buffers->n_gpus * sizeof(uint32_t));
    for (uint32_t i = 0; i < d_buffers->n_gpus; i++) {
        d_buffers->my_gpu_ids[i] = node_rank * d_buffers->n_gpus + i;
        d_buffers->gpus_g2l[d_buffers->my_gpu_ids[i]] = i;
    }
    d_buffers->d_aux_trace = (gl64_gpu **)malloc(deviceCount * sizeof(gl64_gpu*));
    d_buffers->d_constPols = (gl64_gpu **)malloc(deviceCount * sizeof(gl64_gpu*));
    d_buffers->d_constPolsAggregation = (gl64_gpu **)malloc(deviceCount * sizeof(gl64_gpu*));

    for (int i = 0; i < d_buffers->n_gpus; i++) {
        hipSetDevice(d_buffers->my_gpu_ids[i]);
        CHECKCUDAERR(hipMalloc(&d_buffers->d_aux_trace[i], maxSizes->maxAuxTraceArea * sizeof(Goldilocks::Element)));
        CHECKCUDAERR(hipMalloc(&d_buffers->d_constPols[i], maxSizes->totalConstPols * sizeof(Goldilocks::Element)));
        CHECKCUDAERR(hipMalloc(&d_buffers->d_constPolsAggregation[i], maxSizes->totalConstPolsAggregation * sizeof(Goldilocks::Element)));
    }
    init_gpu_const_2(d_buffers->my_gpu_ids, d_buffers->n_gpus);

    TranscriptGL_GPU::init_const(d_buffers->my_gpu_ids, d_buffers->n_gpus);
    return (void *)d_buffers;
}

uint64_t gen_device_streams(void *d_buffers_, uint64_t maxSizeTrace, uint64_t maxSizeContribution, uint64_t maxSizeProverBuffer, uint64_t maxSizeConst, uint64_t maxSizeConstTree, uint64_t maxSizeTraceAggregation, uint64_t maxSizeProverBufferAggregation, uint64_t maxSizeConstAggregation, uint64_t maxSizeConstTreeAggregation, uint64_t maxProofSize, uint64_t maxProofsPerGPU) {
    
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    d_buffers->max_size_prover_buffer = maxSizeProverBuffer;
    d_buffers->max_size_trace = maxSizeTrace;
    d_buffers->max_size_contribution = maxSizeContribution;
    d_buffers->max_size_const = maxSizeConst;
    d_buffers->max_size_const_tree = maxSizeConstTree;
    d_buffers->max_size_trace_aggregation = maxSizeTraceAggregation;
    d_buffers->max_size_prover_buffer_aggregation = maxSizeProverBufferAggregation;
    d_buffers->max_size_const_aggregation = maxSizeConstAggregation;
    d_buffers->max_size_const_tree_aggregation = maxSizeConstTreeAggregation;
    d_buffers->max_size_proof = maxProofSize;
    d_buffers->n_streams = d_buffers->n_gpus * maxProofsPerGPU;
    d_buffers->n_streams_per_gpu = maxProofsPerGPU;

    if (d_buffers->streamsData != nullptr) {
        for (uint64_t i = 0; i < d_buffers->n_streams; i++) {
            d_buffers->streamsData[i].free();
        }
        delete[] d_buffers->streamsData;
    }
    d_buffers->streamsData = new StreamData[d_buffers->n_streams];

    for(uint64_t i=0; i< d_buffers->n_gpus; ++i){
        for (uint64_t j = 0; j < maxProofsPerGPU; j++) {
            d_buffers->streamsData[i*maxProofsPerGPU+j].initialize(maxSizeTrace, maxProofSize, maxSizeConst, maxSizeConstAggregation, maxSizeConstTree, maxSizeConstTreeAggregation, d_buffers->my_gpu_ids[i], j);
        }
    }

    return d_buffers->n_gpus;
}

void free_device_buffers(void *d_buffers_)
{
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;

    for (int i = 0; i < d_buffers->n_gpus; ++i) {
        hipSetDevice(d_buffers->my_gpu_ids[i]);
        CHECKCUDAERR(hipFree(d_buffers->d_aux_trace[i]));
        CHECKCUDAERR(hipFree(d_buffers->d_constPols[i]));
        CHECKCUDAERR(hipFree(d_buffers->d_constPolsAggregation[i]));
    }
    free(d_buffers->d_aux_trace);
    free(d_buffers->d_constPols);
    free(d_buffers->d_constPolsAggregation);
    
    if (d_buffers->streamsData != nullptr) {
        for (uint64_t i = 0; i < d_buffers->n_streams; i++) {
            d_buffers->streamsData[i].free();
        }
        delete[] d_buffers->streamsData;
    }

    for (auto &outer_pair : d_buffers->air_instances) {
        for (auto &inner_pair : outer_pair.second) {
            for (AirInstanceInfo *ptr : inner_pair.second) {
                delete ptr;
            }
        }
    }

    delete d_buffers;
}


void load_device_setup(uint64_t airgroupId, uint64_t airId, char *proofType, void *pSetupCtx_, void *d_buffers_, void *verkeyRoot_) {
    
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    SetupCtx *setupCtx = (SetupCtx *)pSetupCtx_;
    Goldilocks::Element *verkeyRoot = (Goldilocks::Element *)verkeyRoot_;

    std::pair<uint64_t, uint64_t> key = {airgroupId, airId};

    if (d_buffers->air_instances[key][proofType].empty()) {
        d_buffers->air_instances[key][proofType].resize(d_buffers->n_gpus, nullptr);
    }

    for(int i=0; i<d_buffers->n_gpus; ++i){
        hipSetDevice(d_buffers->my_gpu_ids[i]);
        d_buffers->air_instances[key][proofType][i] = new AirInstanceInfo(airgroupId, airId, setupCtx, verkeyRoot);
    }
}

void load_device_const_pols(uint64_t airgroupId, uint64_t airId, uint64_t initial_offset, void *d_buffers_, char *constFilename, uint64_t constSize, char *constTreeFilename, uint64_t constTreeSize, char *proofType) {
    
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    uint64_t sizeConstPols = constSize * sizeof(Goldilocks::Element);
    uint64_t sizeConstTree = constTreeSize * sizeof(Goldilocks::Element);
    
    std::pair<uint64_t, uint64_t> key = {airgroupId, airId};

    uint64_t const_pols_offset = initial_offset;
    uint64_t const_tree_offset = initial_offset + constSize;

    Goldilocks::Element *constPols = new Goldilocks::Element[constSize];
    Goldilocks::Element *constTree = new Goldilocks::Element[constTreeSize];

    loadFileParallel(constPols, constFilename, sizeConstPols);
    loadFileParallel(constTree, constTreeFilename, sizeConstTree);
    
    for(int i=0; i<d_buffers->n_gpus; ++i){
        hipSetDevice(d_buffers->my_gpu_ids[i]);
        gl64_gpu *d_constPols = (strcmp(proofType, "basic") == 0) ? d_buffers->d_constPols[i] : d_buffers->d_constPolsAggregation[i];
        gl64_gpu *d_constTree = (strcmp(proofType, "basic") == 0) ? d_buffers->d_constPols[i] : d_buffers->d_constPolsAggregation[i];
        CHECKCUDAERR(hipMemcpy(d_constPols + const_pols_offset, constPols, sizeConstPols, hipMemcpyHostToDevice));
        CHECKCUDAERR(hipMemcpy(d_constTree + const_tree_offset, constTree, sizeConstTree, hipMemcpyHostToDevice));
        AirInstanceInfo* air_instance_info = d_buffers->air_instances[key][proofType][i];
        air_instance_info->const_pols_offset = const_pols_offset;
        air_instance_info->const_tree_offset = const_tree_offset;
        air_instance_info->stored = true;
    }

    delete[] constPols;
    delete[] constTree;
}

uint64_t gen_proof(void *pSetupCtx_, uint64_t airgroupId, uint64_t airId, uint64_t instanceId, void *params_, void *globalChallenge, uint64_t* proofBuffer, char *proofFile, void *d_buffers_, bool skipRecalculation, uint64_t streamId_, char *constPolsPath,  char *constTreePath) {

    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    uint32_t streamId = skipRecalculation ? streamId_ : selectStream(d_buffers);
    if (skipRecalculation) reserveStream(d_buffers, streamId);
    uint32_t gpuId = d_buffers->streamsData[streamId].gpuId;
    uint32_t gpuLocalId = d_buffers->gpus_g2l[gpuId];
    uint64_t slotId = d_buffers->streamsData[streamId].slotId;
    set_device(gpuId);

    SetupCtx *setupCtx = (SetupCtx *)pSetupCtx_;
    StepsParams *params = (StepsParams *)params_;
    hipStream_t stream = d_buffers->streamsData[streamId].stream;
    TimerGPU &timer = d_buffers->streamsData[streamId].timer;

    gl64_gpu *d_aux_trace = (gl64_gpu *)d_buffers->d_aux_trace[gpuLocalId] + slotId*d_buffers->max_size_prover_buffer;

    uint64_t N = (1 << setupCtx->starkInfo.starkStruct.nBits);
    uint64_t nCols = setupCtx->starkInfo.mapSectionsN["cm1"];
    uint64_t sizeTrace = N * (setupCtx->starkInfo.mapSectionsN["cm1"]) * sizeof(Goldilocks::Element);
    uint64_t sizeConstPols = N * (setupCtx->starkInfo.nConstants) * sizeof(Goldilocks::Element);
    uint64_t sizeConstTree = get_const_tree_size((void *)&setupCtx->starkInfo) * sizeof(Goldilocks::Element);
  
    auto key = std::make_pair(airgroupId, airId);
    std::string proofType = "basic";
    AirInstanceInfo *air_instance_info = d_buffers->air_instances[key][proofType][gpuLocalId];
    uint64_t offset = 0;
    
    if (setupCtx->starkInfo.mapTotalNCustomCommitsFixed > 0) {
        Goldilocks::Element *pCustomCommitsFixed = (Goldilocks::Element *)d_aux_trace + setupCtx->starkInfo.mapOffsets[std::make_pair("custom_fixed", false)];
        CHECKCUDAERR(hipMemcpyAsync(pCustomCommitsFixed, params->pCustomCommitsFixed, setupCtx->starkInfo.mapTotalNCustomCommitsFixed * sizeof(Goldilocks::Element), hipMemcpyHostToDevice, stream));
    }

    if (!air_instance_info->stored && (d_buffers->streamsData[streamId].airgroupId != airgroupId || d_buffers->streamsData[streamId].airId != airId || d_buffers->streamsData[streamId].proofType != "basic")) {
        loadFileParallel(d_buffers->streamsData[streamId].pinned_buffer_const, constPolsPath, sizeConstPols);
        loadFileParallel(d_buffers->streamsData[streamId].pinned_buffer_const_tree, constTreePath, sizeConstTree);
    }

    d_buffers->streamsData[streamId].pSetupCtx = pSetupCtx_;
    d_buffers->streamsData[streamId].proofBuffer = proofBuffer;
    d_buffers->streamsData[streamId].proofFile = string(proofFile);
    d_buffers->streamsData[streamId].airgroupId = airgroupId;
    d_buffers->streamsData[streamId].airId = airId;
    d_buffers->streamsData[streamId].instanceId = instanceId;
    d_buffers->streamsData[streamId].proofType = "basic";

    if (!skipRecalculation) {
        memcpy(d_buffers->streamsData[streamId].pinned_buffer, (Goldilocks::Element *)params->trace, N * nCols * sizeof(Goldilocks::Element));
    }
    offset = N * nCols;
    memcpy(&d_buffers->streamsData[streamId].pinned_buffer[offset], params->publicInputs, setupCtx->starkInfo.nPublics * sizeof(Goldilocks::Element));
    offset += setupCtx->starkInfo.nPublics;
    memcpy(&d_buffers->streamsData[streamId].pinned_buffer[offset], params->proofValues, setupCtx->starkInfo.proofValuesSize * sizeof(Goldilocks::Element));
    offset += setupCtx->starkInfo.proofValuesSize;
    memcpy(&d_buffers->streamsData[streamId].pinned_buffer[offset], params->airgroupValues, setupCtx->starkInfo.airgroupValuesSize * sizeof(Goldilocks::Element));
    offset += setupCtx->starkInfo.airgroupValuesSize;
    memcpy(&d_buffers->streamsData[streamId].pinned_buffer[offset], params->airValues, setupCtx->starkInfo.airValuesSize * sizeof(Goldilocks::Element));
    offset += setupCtx->starkInfo.airValuesSize;
    memcpy(&d_buffers->streamsData[streamId].pinned_buffer[offset], globalChallenge, FIELD_EXTENSION * sizeof(Goldilocks::Element));

    uint64_t offsetStage1 = setupCtx->starkInfo.mapOffsets[std::make_pair("cm1", false)];
    uint64_t offsetPublicInputs = setupCtx->starkInfo.mapOffsets[std::make_pair("publics", false)];
    uint64_t offsetAirgroupValues = setupCtx->starkInfo.mapOffsets[std::make_pair("airgroupvalues", false)];
    uint64_t offsetAirValues = setupCtx->starkInfo.mapOffsets[std::make_pair("airvalues", false)];
    uint64_t offsetProofValues = setupCtx->starkInfo.mapOffsets[std::make_pair("proofvalues", false)];
    uint64_t offsetChallenge = setupCtx->starkInfo.mapOffsets[std::make_pair("challenge", false)];

    offset = 0;
    if (!skipRecalculation) {
        CHECKCUDAERR(hipMemcpyAsync(d_aux_trace + offsetStage1, &d_buffers->streamsData[streamId].pinned_buffer[offset], N * nCols * sizeof(Goldilocks::Element ), hipMemcpyHostToDevice, stream));
    }
    offset += N * nCols;
    CHECKCUDAERR(hipMemcpyAsync(d_aux_trace + offsetPublicInputs, &d_buffers->streamsData[streamId].pinned_buffer[offset], setupCtx->starkInfo.nPublics * sizeof(Goldilocks::Element), hipMemcpyHostToDevice, stream));
    offset += setupCtx->starkInfo.nPublics;
    if (setupCtx->starkInfo.proofValuesSize > 0) {
        CHECKCUDAERR(hipMemcpyAsync(d_aux_trace + offsetProofValues, &d_buffers->streamsData[streamId].pinned_buffer[offset], setupCtx->starkInfo.proofValuesSize * sizeof(Goldilocks::Element), hipMemcpyHostToDevice, stream));
        offset += setupCtx->starkInfo.proofValuesSize;
    }
    if (setupCtx->starkInfo.airgroupValuesSize > 0) {
        CHECKCUDAERR(hipMemcpyAsync(d_aux_trace + offsetAirgroupValues, &d_buffers->streamsData[streamId].pinned_buffer[offset], setupCtx->starkInfo.airgroupValuesSize * sizeof(Goldilocks::Element), hipMemcpyHostToDevice, stream));
        offset += setupCtx->starkInfo.airgroupValuesSize;
    }
    if (setupCtx->starkInfo.airValuesSize > 0) {
        CHECKCUDAERR(hipMemcpyAsync(d_aux_trace + offsetAirValues, &d_buffers->streamsData[streamId].pinned_buffer[offset], setupCtx->starkInfo.airValuesSize * sizeof(Goldilocks::Element), hipMemcpyHostToDevice, stream));
        offset += setupCtx->starkInfo.airValuesSize;
    }
    CHECKCUDAERR(hipMemcpyAsync(d_aux_trace + offsetChallenge, &d_buffers->streamsData[streamId].pinned_buffer[offset], FIELD_EXTENSION * sizeof(Goldilocks::Element), hipMemcpyHostToDevice, stream));
    offset += FIELD_EXTENSION;

    gl64_gpu *d_const_pols;
    gl64_gpu *d_const_tree;
    if (air_instance_info->stored) {
        d_const_pols = d_buffers->d_constPols[gpuLocalId] + air_instance_info->const_pols_offset;
        d_const_tree = d_buffers->d_constPols[gpuLocalId] + air_instance_info->const_tree_offset;
    } else {
        uint64_t offsetConstTree = setupCtx->starkInfo.mapOffsets[std::make_pair("const", true)];
        uint64_t offsetConstPols = setupCtx->starkInfo.mapOffsets[std::make_pair("const", false)];
        CHECKCUDAERR(hipMemcpyAsync(d_aux_trace + offsetConstPols, d_buffers->streamsData[streamId].pinned_buffer_const, sizeConstPols, hipMemcpyHostToDevice, stream));
        CHECKCUDAERR(hipMemcpyAsync(d_aux_trace + offsetConstTree, d_buffers->streamsData[streamId].pinned_buffer_const_tree, sizeConstTree, hipMemcpyHostToDevice, stream));
        d_const_pols = d_aux_trace + offsetConstPols;
        d_const_tree = d_aux_trace + offsetConstTree;
    }


    genProof_gpu(*setupCtx, d_aux_trace, d_const_pols, d_const_tree, streamId, d_buffers, air_instance_info, skipRecalculation, timer, stream);
    hipEventRecord(d_buffers->streamsData[streamId].end_event, stream);
    d_buffers->streamsData[streamId].status = 2;
    return streamId;


}

void get_proof(DeviceCommitBuffers *d_buffers, uint64_t streamId) {

    uint32_t gpuId = d_buffers->streamsData[streamId].gpuId;
    set_device(gpuId);

    SetupCtx *setupCtx = (SetupCtx*) d_buffers->streamsData[streamId].pSetupCtx;
    uint64_t airgroupId = d_buffers->streamsData[streamId].airgroupId;
    uint64_t airId = d_buffers->streamsData[streamId].airId;
    uint64_t instanceId = d_buffers->streamsData[streamId].instanceId;
    uint64_t * proofBuffer = d_buffers->streamsData[streamId].proofBuffer;
    string proofType = d_buffers->streamsData[streamId].proofType;
    string proofFile = d_buffers->streamsData[streamId].proofFile;
    TimerGPU &timer = d_buffers->streamsData[streamId].timer;

    closeStreamTimer(timer, true);

    writeProof(*setupCtx, d_buffers->streamsData[streamId].pinned_buffer_proof, proofBuffer, airgroupId, airId, instanceId, proofFile);

    if (proof_done_callback != nullptr) {
        proof_done_callback(instanceId, proofType.c_str());
    }
}

void get_stream_proofs(void *d_buffers_){
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    std::lock_guard<std::mutex> lock(d_buffers->mutex_slot_selection);
    for (uint64_t i = 0; i < d_buffers->n_streams; i++) {
        if (d_buffers->streamsData[i].status == 0 || d_buffers->streamsData[i].status == 3) continue;
        set_device(d_buffers->streamsData[i].gpuId);
        CHECKCUDAERR(hipStreamSynchronize(d_buffers->streamsData[i].stream));
        if(d_buffers->streamsData[i].root != nullptr) {
            get_commit_root(d_buffers, i);
        }else{
            get_proof(d_buffers, i);
        }
        d_buffers->streamsData[i].reset();        
    }
}

void get_stream_proofs_non_blocking(void *d_buffers_){
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    std::lock_guard<std::mutex> lock(d_buffers->mutex_slot_selection);
    for (uint64_t i = 0; i < d_buffers->n_streams; i++) {
        if(d_buffers->streamsData[i].status==2 &&  hipEventQuery(d_buffers->streamsData[i].end_event) == hipSuccess){
            set_device(d_buffers->streamsData[i].gpuId);
            if(d_buffers->streamsData[i].root != nullptr) {
                get_commit_root(d_buffers, i);
            }else{
                get_proof(d_buffers, i);
            }
            d_buffers->streamsData[i].reset();        
        }
    }
}

void get_stream_id_proof(void *d_buffers_, uint64_t streamId) {
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    set_device(d_buffers->streamsData[streamId].gpuId);
    CHECKCUDAERR(hipStreamSynchronize(d_buffers->streamsData[streamId].stream));
    if(d_buffers->streamsData[streamId].root != nullptr) {
            get_commit_root(d_buffers, streamId);
        }else{
            get_proof(d_buffers, streamId);
        }
    d_buffers->streamsData[streamId].reset();
}

uint64_t gen_recursive_proof(void *pSetupCtx_, char *globalInfoFile, uint64_t airgroupId, uint64_t airId, uint64_t instanceId, void *trace, void *aux_trace, void *pConstPols, void *pConstTree, void *pPublicInputs, uint64_t* proofBuffer, char *proof_file, bool vadcop, void *d_buffers_, char *constPolsPath, char *constTreePath, char *proofType)
{
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    uint32_t streamId = selectStream(d_buffers);
    uint32_t gpuId = d_buffers->streamsData[streamId].gpuId;
    uint32_t gpuLocalId = d_buffers->gpus_g2l[gpuId];
    uint64_t slotId =  d_buffers->streamsData[streamId].slotId;
    set_device(gpuId);

    SetupCtx *setupCtx = (SetupCtx *)pSetupCtx_;
    hipStream_t stream = d_buffers->streamsData[streamId].stream;
    TimerGPU &timer = d_buffers->streamsData[streamId].timer;
    
    gl64_gpu *d_trace = (gl64_gpu *)d_buffers->d_aux_trace[gpuLocalId] + slotId*d_buffers->max_size_prover_buffer;
    gl64_gpu *d_aux_trace = d_trace + d_buffers->max_size_trace_aggregation;

    uint64_t N = (1 << setupCtx->starkInfo.starkStruct.nBits);
    uint64_t nCols = setupCtx->starkInfo.mapSectionsN["cm1"];
    uint64_t sizeTrace = N * (setupCtx->starkInfo.mapSectionsN["cm1"]) * sizeof(Goldilocks::Element);
    uint64_t sizeConstPols = N * (setupCtx->starkInfo.nConstants) * sizeof(Goldilocks::Element);
    uint64_t sizeConstTree = get_const_tree_size((void *)&setupCtx->starkInfo) * sizeof(Goldilocks::Element);

    auto key = std::make_pair(airgroupId, airId);
    AirInstanceInfo *air_instance_info = d_buffers->air_instances[key][string(proofType)][gpuLocalId];

    memcpy(d_buffers->streamsData[streamId].pinned_buffer, (Goldilocks::Element *)trace, N * nCols * sizeof(Goldilocks::Element));
    if (!air_instance_info->stored && (d_buffers->streamsData[streamId].airgroupId != airgroupId || d_buffers->streamsData[streamId].airId != airId || d_buffers->streamsData[streamId].proofType != string(proofType))) {
        loadFileParallel(d_buffers->streamsData[streamId].pinned_buffer_const, constPolsPath, sizeConstPols);
        loadFileParallel(d_buffers->streamsData[streamId].pinned_buffer_const_tree, constTreePath, sizeConstTree);
    }

    d_buffers->streamsData[streamId].pSetupCtx = pSetupCtx_;
    d_buffers->streamsData[streamId].proofBuffer = proofBuffer;
    d_buffers->streamsData[streamId].proofFile = string(proof_file);
    d_buffers->streamsData[streamId].airgroupId = airgroupId;
    d_buffers->streamsData[streamId].airId = airId;
    d_buffers->streamsData[streamId].instanceId = instanceId;
    d_buffers->streamsData[streamId].proofType = string(proofType);

    memcpy(&d_buffers->streamsData[streamId].pinned_buffer[N * nCols], (Goldilocks::Element *)pPublicInputs, setupCtx->starkInfo.nPublics * sizeof(Goldilocks::Element));

    uint64_t offsetPublicInputs = setupCtx->starkInfo.mapOffsets[std::make_pair("publics", false)];
    CHECKCUDAERR(hipMemcpyAsync(d_trace, d_buffers->streamsData[streamId].pinned_buffer, sizeTrace, hipMemcpyHostToDevice, stream));
    CHECKCUDAERR(hipMemcpyAsync(d_aux_trace + offsetPublicInputs, &d_buffers->streamsData[streamId].pinned_buffer[N * nCols], setupCtx->starkInfo.nPublics * sizeof(Goldilocks::Element), hipMemcpyHostToDevice, stream));
    
    gl64_gpu *d_const_pols;
    gl64_gpu *d_const_tree;
    if (air_instance_info->stored) {
        d_const_pols = d_buffers->d_constPolsAggregation[gpuLocalId] + air_instance_info->const_pols_offset;
        d_const_tree = d_buffers->d_constPolsAggregation[gpuLocalId] + air_instance_info->const_tree_offset;
    } else {
        uint64_t offsetConstTree = setupCtx->starkInfo.mapOffsets[std::make_pair("const", true)];
        uint64_t offsetConstPols = setupCtx->starkInfo.mapOffsets[std::make_pair("const", false)];
        d_const_pols = d_aux_trace + offsetConstPols;
        d_const_tree = d_aux_trace + offsetConstTree;
        CHECKCUDAERR(hipMemcpyAsync(d_const_pols, d_buffers->streamsData[streamId].pinned_buffer_const, sizeConstPols, hipMemcpyHostToDevice, stream));
        CHECKCUDAERR(hipMemcpyAsync(d_const_tree, d_buffers->streamsData[streamId].pinned_buffer_const_tree, sizeConstTree, hipMemcpyHostToDevice, stream));
    }

    genRecursiveProof_gpu<Goldilocks::Element>(*setupCtx, d_trace, d_aux_trace, d_const_pols, d_const_tree, streamId, d_buffers, air_instance_info, instanceId, timer, stream);
    hipEventRecord(d_buffers->streamsData[streamId].end_event, stream);
    d_buffers->streamsData[streamId].status = 2;
    return streamId;
}

uint64_t commit_witness(uint64_t arity, uint64_t nBits, uint64_t nBitsExt, uint64_t nCols, uint64_t instanceId, void *root, void *trace, void *auxTrace, void *d_buffers_, void *pSetupCtx_) {

    SetupCtx *setupCtx = (SetupCtx *)pSetupCtx_;
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    uint32_t streamId = selectStream(d_buffers);
    uint32_t gpuId = d_buffers->streamsData[streamId].gpuId;
    uint32_t gpuLocalId = d_buffers->gpus_g2l[gpuId];
    uint64_t slotId = d_buffers->streamsData[streamId].slotId;
    set_device(gpuId);

    d_buffers->streamsData[streamId].root = root;
    d_buffers->streamsData[streamId].instanceId = instanceId;

    uint64_t N = 1 << nBits;

    hipStream_t stream = d_buffers->streamsData[streamId].stream;
    TimerGPU &timer = d_buffers->streamsData[streamId].timer;

    gl64_gpu *d_aux_trace = (gl64_gpu *)d_buffers->d_aux_trace[gpuLocalId] + slotId*d_buffers->max_size_prover_buffer;
    uint64_t sizeTrace = N * nCols * sizeof(Goldilocks::Element);
    uint64_t offsetStage1 = setupCtx->starkInfo.mapOffsets[std::make_pair("cm1", false)];

    memcpy(d_buffers->streamsData[streamId].pinned_buffer, (Goldilocks::Element *)trace, N * nCols * sizeof(Goldilocks::Element));
    CHECKCUDAERR(hipMemcpyAsync(d_aux_trace + offsetStage1, d_buffers->streamsData[streamId].pinned_buffer, sizeTrace, hipMemcpyHostToDevice, stream));
    genCommit_gpu(arity, nBits, nBitsExt, nCols, d_aux_trace, d_buffers->streamsData[streamId].pinned_buffer_proof, setupCtx, timer, stream);

    hipEventRecord(d_buffers->streamsData[streamId].end_event, stream);
    d_buffers->streamsData[streamId].status = 2;
    return streamId;
}

void get_commit_root(DeviceCommitBuffers *d_buffers, uint64_t streamId) {

    set_device(d_buffers->streamsData[streamId].gpuId);
    Goldilocks::Element *root = (Goldilocks::Element *)d_buffers->streamsData[streamId].root;
    memcpy((Goldilocks::Element *)root, d_buffers->streamsData[streamId].pinned_buffer_proof, HASH_SIZE * sizeof(uint64_t));
    closeStreamTimer(d_buffers->streamsData[streamId].timer, false);
    
    uint64_t instanceId = d_buffers->streamsData[streamId].instanceId;

    if (proof_done_callback != nullptr) {
        proof_done_callback(instanceId, "");
    }

}

uint64_t check_device_memory() {
    
    set_device(0); //We assume that all the GPUs have the same characteristics, we only check the GPU 0
    uint64_t freeMem, totalMem;
    hipError_t err = hipMemGetInfo(&freeMem, &totalMem);
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return 0;
    }

    std::cout << "Free memory: " << freeMem / (1024.0 * 1024.0) << " MB" << std::endl;
    std::cout << "Total memory: " << totalMem / (1024.0 * 1024.0) << " MB" << std::endl;

    return freeMem;
}

// Function to set the CUDA device based on the MPI rank
void set_device_mpi(uint32_t mpi_node_rank){
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found." << std::endl;
        exit(1);
    }
    int device = mpi_node_rank % deviceCount;
    hipSetDevice(device);
}

void set_device(uint32_t gpuId){
    hipSetDevice(gpuId);
}

uint32_t selectStream(DeviceCommitBuffers* d_buffers){

    std::lock_guard<std::mutex> lock(d_buffers->mutex_slot_selection);
    uint32_t countFreeStreamsGPU[d_buffers->n_gpus];
    uint32_t countUnusedStreams[d_buffers->n_gpus];
    int streamIdxGPU[d_buffers->n_gpus];
    
    for( uint32_t i = 0; i < d_buffers->n_gpus; i++){
        countUnusedStreams[i] = 0;
        countFreeStreamsGPU[i] = 0;
        streamIdxGPU[i] = -1;
    }

    bool someFree = false;
    while (!someFree){
        for (uint32_t i = 0; i < d_buffers->n_streams; i++) {
            if (d_buffers->streamsData[i].status==0 || d_buffers->streamsData[i].status==3 || (d_buffers->streamsData[i].status==2 &&  hipEventQuery(d_buffers->streamsData[i].end_event) == hipSuccess)) {

                countFreeStreamsGPU[d_buffers->gpus_g2l[d_buffers->streamsData[i].gpuId]]++;
                if(d_buffers->streamsData[i].status==0){
                    countUnusedStreams[d_buffers->gpus_g2l[d_buffers->streamsData[i].gpuId]]++;
                    streamIdxGPU[d_buffers->gpus_g2l[d_buffers->streamsData[i].gpuId]] = i;
                }
                if( streamIdxGPU[d_buffers->gpus_g2l[d_buffers->streamsData[i].gpuId]] == -1 ){
                    streamIdxGPU[d_buffers->gpus_g2l[d_buffers->streamsData[i].gpuId]] = i;
                }
                someFree = true;
            }
        }
        if (!someFree)
            std::this_thread::sleep_for(std::chrono::microseconds(300)); 
    }

    uint32_t maxFree = 0;
    uint32_t streamId = 0;
    for (uint32_t i = 0; i < d_buffers->n_gpus; i++) {
        if (countFreeStreamsGPU[i] > maxFree || (countFreeStreamsGPU[i] == maxFree && countUnusedStreams[i] > countUnusedStreams[streamId])) {
            maxFree = countFreeStreamsGPU[i];
            streamId = streamIdxGPU[i];
        }
    }

    reserveStream(d_buffers, streamId);
    return streamId;
}

void reserveStream(DeviceCommitBuffers* d_buffers, uint32_t streamId){
    if(d_buffers->streamsData[streamId].status==2 &&  hipEventQuery(d_buffers->streamsData[streamId].end_event) == hipSuccess) {

        if(d_buffers->streamsData[streamId].root != nullptr) {
            get_commit_root(d_buffers, streamId);
        }else{
            get_proof(d_buffers, streamId);
        }
        d_buffers->streamsData[streamId].reset();
    }

    d_buffers->streamsData[streamId].status = 1;
}

void closeStreamTimer(TimerGPU &timer, bool isProve) {
    TimerSyncAndLogAllGPU(timer); 
    TimerSyncCategoriesGPU(timer);
    if(isProve)
        TimerLogCategoryContributionsGPU(timer, STARK_GPU_PROOF);
    else
        TimerLogCategoryContributionsGPU(timer, STARK_GPU_COMMIT);
    TimerResetGPU(timer);
}
#endif