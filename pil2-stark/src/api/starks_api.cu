#include "zkglobals.hpp"
#include "proof2zkinStark.hpp"
#include "starks.hpp"
#include "omp.h"
#include "starks_api.hpp"
#ifdef __USE_CUDA__
#include "gen_recursive_proof.cuh"
#include "gen_proof.cuh"
#include "gen_commit.cuh"

struct MaxSizes
{
    uint64_t maxN;
    uint64_t maxNExtended;
    uint64_t maxTraceArea;
    uint64_t maxNTTArea;
    uint64_t maxConstArea;
    uint64_t maxNPublics;
    uint64_t maxAuxTraceArea;
    uint64_t maxConstTreeSize;
};


void *gen_device_commit_buffers(void *maxSizes_)
{
    MaxSizes *maxSizes = (MaxSizes *)maxSizes_;
    CHECKCUDAERR(hipSetDevice(0));
    DeviceCommitBuffers *buffers = new DeviceCommitBuffers();
    CHECKCUDAERR(hipMalloc(&buffers->d_trace, maxSizes->maxTraceArea * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_constPols, maxSizes->maxConstArea * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_constTree, maxSizes->maxConstTreeSize * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_publicInputs, maxSizes->maxNPublics * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_aux_trace, maxSizes->maxAuxTraceArea * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_forwardTwiddleFactors, maxSizes->maxNExtended * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_inverseTwiddleFactors, maxSizes->maxNExtended * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_r, maxSizes->maxNExtended * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_ntt, maxSizes->maxNTTArea * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_tree, maxSizes->maxNExtended * sizeof(uint64_t)));
    return (void *)buffers;

}

void gen_proof(void *pSetupCtx_, uint64_t airgroupId, uint64_t airId, uint64_t instanceId, void *params_, void *globalChallenge, uint64_t* proofBuffer, char *proofFile, void *d_buffers_) {

    double time = omp_get_wtime();
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    SetupCtx *setupCtx = (SetupCtx *)pSetupCtx_;
    StepsParams *params = (StepsParams *)params_;

    uint64_t N = (1 << setupCtx->starkInfo.starkStruct.nBits);
    uint64_t sizeTrace = N * (setupCtx->starkInfo.mapSectionsN["cm1"]) * sizeof(Goldilocks::Element);
    uint64_t sizeConstPols = N * (setupCtx->starkInfo.nConstants) * sizeof(Goldilocks::Element);
    uint64_t sizeConstTree = get_const_tree_size((void *)&setupCtx->starkInfo) * sizeof(Goldilocks::Element);

    CHECKCUDAERR(hipSetDevice(0));
    CHECKCUDAERR(hipMemcpy(d_buffers->d_trace, params->trace, sizeTrace, hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(d_buffers->d_constPols, params->pConstPolsAddress, sizeConstPols, hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(d_buffers->d_constTree, params->pConstPolsExtendedTreeAddress, sizeConstTree, hipMemcpyHostToDevice));

    time = omp_get_wtime() - time;
    //std::cout << "rick genDeviceBuffers time: " << time << std::endl;

    time = omp_get_wtime();
    genProof_gpu(*setupCtx, airgroupId, airId, instanceId, *params, (Goldilocks::Element *)globalChallenge, proofBuffer, string(proofFile), d_buffers);
    time = omp_get_wtime() - time;
    //std::cout << "rick genRecursiveProof_gpu time: " << time << std::endl;
}

void gen_recursive_proof(void *pSetupCtx_, char *globalInfoFile, uint64_t airgroupId, uint64_t airId, uint64_t instanceId, void *trace, void *aux_trace, void *pConstPols, void *pConstTree, void *pPublicInputs, uint64_t* proofBuffer, char *proof_file, bool vadcop, void *d_buffers_)
{

    json globalInfo;
    file2json(globalInfoFile, globalInfo);

    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    SetupCtx *setupCtx = (SetupCtx *)pSetupCtx_;
    double time = omp_get_wtime();

    CHECKCUDAERR(hipSetDevice(0));
    uint64_t N = (1 << setupCtx->starkInfo.starkStruct.nBits);
    uint64_t sizeTrace = N * (setupCtx->starkInfo.mapSectionsN["cm1"]) * sizeof(Goldilocks::Element);
    uint64_t sizeConstPols = N * (setupCtx->starkInfo.nConstants) * sizeof(Goldilocks::Element);
    uint64_t sizeConstTree = get_const_tree_size((void *)&setupCtx->starkInfo) * sizeof(Goldilocks::Element);

    CHECKCUDAERR(hipMemcpy(d_buffers->d_trace, trace, sizeTrace, hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(d_buffers->d_constPols, pConstPols, sizeConstPols, hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(d_buffers->d_constTree, pConstTree, sizeConstTree, hipMemcpyHostToDevice));

    time = omp_get_wtime() - time;
    std::cout << "rick genDeviceBuffers time: " << time << std::endl;

    time = omp_get_wtime();
    genRecursiveProof_gpu<Goldilocks::Element>(*setupCtx, globalInfo, airgroupId, airId, instanceId, (Goldilocks::Element *)trace, (Goldilocks::Element *)pConstPols, (Goldilocks::Element *)pConstTree, (Goldilocks::Element *)pPublicInputs, proofBuffer, string(proof_file), d_buffers, vadcop);
    time = omp_get_wtime() - time;
    std::cout << "rick genRecursiveProof_gpu time: " << time << std::endl;
}

void commit_witness(uint64_t arity, uint64_t nBits, uint64_t nBitsExt, uint64_t nCols, void *root, void *trace, void *auxTrace, void *d_buffers_) {

    double time = omp_get_wtime();

    Goldilocks::Element *rootGL = (Goldilocks::Element *)root;
    uint64_t N = 1 << nBits;
    uint64_t NExtended = 1 << nBitsExt;


    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    uint64_t sizeTrace = N * nCols * sizeof(Goldilocks::Element);
    CHECKCUDAERR(hipSetDevice(0));
    CHECKCUDAERR(hipMemcpy(d_buffers->d_trace, trace, sizeTrace, hipMemcpyHostToDevice));
    genCommit_gpu(arity, rootGL, N, NExtended, nCols, d_buffers);
    time = omp_get_wtime() - time;
    //std::cout << "rick genRCommit_gpu time: " << time << std::endl;
}
#endif