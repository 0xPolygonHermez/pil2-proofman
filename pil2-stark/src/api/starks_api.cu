#include "hip/hip_runtime.h"
#include "zkglobals.hpp"
#include "proof2zkinStark.hpp"
#include "starks.hpp"
#include "omp.h"
#include "starks_api.hpp"
#include "starks_api_internal.hpp"
#include <cstring>
#include <thread>

#ifdef __USE_CUDA__
#include "gen_recursive_proof.cuh"
#include "gen_proof.cuh"
#include "gen_commit.cuh"
#include "poseidon2_goldilocks.cu"
#include <hip/hip_runtime.h>
#include <mutex>


struct MaxSizes
{
    uint64_t totalConstPols;
    uint64_t maxAuxTraceArea;
    uint64_t totalConstPolsAggregation;
};

uint32_t selectStream(DeviceCommitBuffers* d_buffers);
void reserveStream(DeviceCommitBuffers* d_buffers, uint32_t streamId);

void closeStreamTimer(TimerGPU &timer, bool isProve);
void get_proof(DeviceCommitBuffers *d_buffers, uint64_t streamId);
void get_commit_root(DeviceCommitBuffers *d_buffers, uint64_t streamId);



void *gen_device_buffers(void *maxSizes_, uint32_t node_rank, uint32_t node_size)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    MaxSizes *maxSizes = (MaxSizes *)maxSizes_;


    if(deviceCount >= node_size) {
       
        if (deviceCount % node_size != 0) {
            zklog.error("Device count must be divisible by number of processes per node");
            exit(1);
        }
        
        DeviceCommitBuffers *d_buffers = new DeviceCommitBuffers();
        d_buffers->n_gpus = (uint32_t) deviceCount / node_size;
        d_buffers->gpus_g2l = (uint32_t *)malloc(deviceCount * sizeof(uint32_t));
        d_buffers->my_gpu_ids = (uint32_t *)malloc(d_buffers->n_gpus * sizeof(uint32_t));
        for (uint32_t i = 0; i < d_buffers->n_gpus; i++) {
            d_buffers->my_gpu_ids[i] = node_rank * d_buffers->n_gpus + i;
            d_buffers->gpus_g2l[d_buffers->my_gpu_ids[i]] = i;
        }
        d_buffers->d_aux_trace = (gl64_t **)malloc(d_buffers->n_gpus * sizeof(gl64_t*));
        d_buffers->d_constPols = (gl64_t **)malloc(d_buffers->n_gpus * sizeof(gl64_t*));
        d_buffers->d_constPolsAggregation = (gl64_t **)malloc(d_buffers->n_gpus * sizeof(gl64_t*));

        for (int i = 0; i < d_buffers->n_gpus; i++) {
            hipSetDevice(d_buffers->my_gpu_ids[i]);
            CHECKCUDAERR(hipMalloc(&d_buffers->d_aux_trace[i], maxSizes->maxAuxTraceArea * sizeof(Goldilocks::Element)));
            CHECKCUDAERR(hipMalloc(&d_buffers->d_constPols[i], maxSizes->totalConstPols * sizeof(Goldilocks::Element)));
            CHECKCUDAERR(hipMalloc(&d_buffers->d_constPolsAggregation[i], maxSizes->totalConstPolsAggregation * sizeof(Goldilocks::Element)));
        }
        init_gpu_const_2(d_buffers->my_gpu_ids, d_buffers->n_gpus);

        TranscriptGL_GPU::init_const(d_buffers->my_gpu_ids, d_buffers->n_gpus);


#ifdef NUMA_NODE
        // Check device afinity with process NUMA node
        for (int i = 0; i < d_buffers->n_gpus; i++) {
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, d_buffers->my_gpu_ids[i]);
            if (prop.numaNode == -1) {
                zklog.warning("Cannot verify NUMA affinity: GPU %d's NUMA node is unknown (prop.numaNode == -1). "
                            "Assuming it matches process NUMA node %d", 
                            d_buffers->my_gpu_ids[i], NUMA_NODE);
            } 
            else if (prop.numaNode != NUMA_NODE) {
                zklog.error("NUMA affinity violation: GPU %d is on NUMA node %d, but process is bound to NUMA node %d",
                        d_buffers->my_gpu_ids[i], prop.numaNode, NUMA_NODE);
                exit(1);
            }
            else {
                zklog.info("Verified GPU %d is on correct NUMA node %d", 
                        d_buffers->my_gpu_ids[i], NUMA_NODE);
            }
        }
#endif
        return (void *)d_buffers;
    } else {

        if (node_size % deviceCount  != 0) {
            zklog.error("Number of processes per node must be divisible by device count");
            exit(1);
        }
        
        DeviceCommitBuffers *d_buffers = new DeviceCommitBuffers();
        d_buffers->n_gpus = 1;
        d_buffers->gpus_g2l = (uint32_t *)malloc(deviceCount * sizeof(uint32_t));
        d_buffers->my_gpu_ids = (uint32_t *)malloc(d_buffers->n_gpus * sizeof(uint32_t));
        d_buffers->my_gpu_ids[0] = node_rank % deviceCount;
        d_buffers->gpus_g2l[d_buffers->my_gpu_ids[0]] = 0;
        
        d_buffers->d_aux_trace = (gl64_t **)malloc(d_buffers->n_gpus * sizeof(gl64_t*));
        d_buffers->d_constPols = (gl64_t **)malloc(d_buffers->n_gpus * sizeof(gl64_t*));
        d_buffers->d_constPolsAggregation = (gl64_t **)malloc(d_buffers->n_gpus * sizeof(gl64_t*));

        hipSetDevice(d_buffers->my_gpu_ids[0]);
        CHECKCUDAERR(hipMalloc(&d_buffers->d_aux_trace[0], maxSizes->maxAuxTraceArea * sizeof(Goldilocks::Element)));
        CHECKCUDAERR(hipMalloc(&d_buffers->d_constPols[0], maxSizes->totalConstPols * sizeof(Goldilocks::Element)));
        CHECKCUDAERR(hipMalloc(&d_buffers->d_constPolsAggregation[0], maxSizes->totalConstPolsAggregation * sizeof(Goldilocks::Element)));
        
        init_gpu_const_2(d_buffers->my_gpu_ids, d_buffers->n_gpus);

        TranscriptGL_GPU::init_const(d_buffers->my_gpu_ids, d_buffers->n_gpus);
        return (void *)d_buffers;
    }
}

uint64_t gen_device_streams(void *d_buffers_, uint64_t maxSizeTrace, uint64_t maxSizeContribution, uint64_t maxSizeProverBuffer, uint64_t maxSizeConst, uint64_t maxSizeConstTree, uint64_t maxSizeTraceAggregation, uint64_t maxSizeProverBufferAggregation, uint64_t maxSizeConstAggregation, uint64_t maxSizeConstTreeAggregation, uint64_t maxProofSize, uint64_t maxProofsPerGPU) {
    
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    d_buffers->max_size_prover_buffer = maxSizeProverBuffer;
    d_buffers->max_size_trace = maxSizeTrace;
    d_buffers->max_size_contribution = maxSizeContribution;
    d_buffers->max_size_const = maxSizeConst;
    d_buffers->max_size_const_tree = maxSizeConstTree;
    d_buffers->max_size_trace_aggregation = maxSizeTraceAggregation;
    d_buffers->max_size_prover_buffer_aggregation = maxSizeProverBufferAggregation;
    d_buffers->max_size_const_aggregation = maxSizeConstAggregation;
    d_buffers->max_size_const_tree_aggregation = maxSizeConstTreeAggregation;
    d_buffers->max_size_proof = maxProofSize;
    d_buffers->n_streams = d_buffers->n_gpus * maxProofsPerGPU;
    d_buffers->n_streams_per_gpu = maxProofsPerGPU;

    if (d_buffers->streamsData != nullptr) {
        for (uint64_t i = 0; i < d_buffers->n_streams; i++) {
            d_buffers->streamsData[i].free();
        }
        delete[] d_buffers->streamsData;
    }
    d_buffers->streamsData = new StreamData[d_buffers->n_streams];

    for(uint64_t i=0; i< d_buffers->n_gpus; ++i){
        for (uint64_t j = 0; j < maxProofsPerGPU; j++) {
            d_buffers->streamsData[j*d_buffers->n_gpus+i].initialize(maxSizeTrace, maxProofSize, maxSizeConst, maxSizeConstAggregation, maxSizeConstTree, maxSizeConstTreeAggregation, d_buffers->my_gpu_ids[i], j);
        }
    }

    return d_buffers->n_gpus;
}

void free_device_buffers(void *d_buffers_)
{
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;

    for (int i = 0; i < d_buffers->n_gpus; ++i) {
        hipSetDevice(d_buffers->my_gpu_ids[i]);
        CHECKCUDAERR(hipFree(d_buffers->d_aux_trace[i]));
        CHECKCUDAERR(hipFree(d_buffers->d_constPols[i]));
        CHECKCUDAERR(hipFree(d_buffers->d_constPolsAggregation[i]));
    }
    free(d_buffers->d_aux_trace);
    free(d_buffers->d_constPols);
    free(d_buffers->d_constPolsAggregation);
    
    if (d_buffers->streamsData != nullptr) {
        for (uint64_t i = 0; i < d_buffers->n_streams; i++) {
            d_buffers->streamsData[i].free();
        }
        delete[] d_buffers->streamsData;
    }

    for (auto &outer_pair : d_buffers->air_instances) {
        for (auto &inner_pair : outer_pair.second) {
            for (AirInstanceInfo *ptr : inner_pair.second) {
                delete ptr;
            }
        }
    }

    delete d_buffers;
}


void load_device_setup(uint64_t airgroupId, uint64_t airId, char *proofType, void *pSetupCtx_, void *d_buffers_, void *verkeyRoot_) {
    
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    SetupCtx *setupCtx = (SetupCtx *)pSetupCtx_;
    Goldilocks::Element *verkeyRoot = (Goldilocks::Element *)verkeyRoot_;

    std::pair<uint64_t, uint64_t> key = {airgroupId, airId};

    if (d_buffers->air_instances[key][proofType].empty()) {
        d_buffers->air_instances[key][proofType].resize(d_buffers->n_gpus, nullptr);
    }

    for(int i=0; i<d_buffers->n_gpus; ++i){
        hipSetDevice(d_buffers->my_gpu_ids[i]);
        d_buffers->air_instances[key][proofType][i] = new AirInstanceInfo(airgroupId, airId, setupCtx, verkeyRoot);
    }
}

void load_device_const_pols(uint64_t airgroupId, uint64_t airId, uint64_t initial_offset, void *d_buffers_, char *constFilename, uint64_t constSize, char *constTreeFilename, uint64_t constTreeSize, char *proofType) {
    
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    uint64_t sizeConstPols = constSize * sizeof(Goldilocks::Element);
    uint64_t sizeConstTree = constTreeSize * sizeof(Goldilocks::Element);
    
    std::pair<uint64_t, uint64_t> key = {airgroupId, airId};

    uint64_t const_pols_offset = initial_offset;
    uint64_t const_tree_offset = initial_offset + constSize;

    Goldilocks::Element *constPols = new Goldilocks::Element[constSize];
    Goldilocks::Element *constTree = new Goldilocks::Element[constTreeSize];

    loadFileParallel(constPols, constFilename, sizeConstPols);
    loadFileParallel(constTree, constTreeFilename, sizeConstTree);
    
    for(int i=0; i<d_buffers->n_gpus; ++i){
        hipSetDevice(d_buffers->my_gpu_ids[i]);
        gl64_t *d_constPols = (strcmp(proofType, "basic") == 0) ? d_buffers->d_constPols[i] : d_buffers->d_constPolsAggregation[i];
        gl64_t *d_constTree = (strcmp(proofType, "basic") == 0) ? d_buffers->d_constPols[i] : d_buffers->d_constPolsAggregation[i];
        CHECKCUDAERR(hipMemcpy(d_constPols + const_pols_offset, constPols, sizeConstPols, hipMemcpyHostToDevice));
        CHECKCUDAERR(hipMemcpy(d_constTree + const_tree_offset, constTree, sizeConstTree, hipMemcpyHostToDevice));
        AirInstanceInfo* air_instance_info = d_buffers->air_instances[key][proofType][i];
        air_instance_info->const_pols_offset = const_pols_offset;
        air_instance_info->const_tree_offset = const_tree_offset;
        air_instance_info->stored = true;
    }

    delete[] constPols;
    delete[] constTree;
}

uint64_t gen_proof(void *pSetupCtx_, uint64_t airgroupId, uint64_t airId, uint64_t instanceId, void *params_, void *globalChallenge, uint64_t* proofBuffer, char *proofFile, void *d_buffers_, bool skipRecalculation, uint64_t streamId_, char *constPolsPath,  char *constTreePath) {

    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    uint32_t streamId = skipRecalculation ? streamId_ : selectStream(d_buffers);
    if (skipRecalculation) reserveStream(d_buffers, streamId);
    uint32_t gpuId = d_buffers->streamsData[streamId].gpuId;
    uint32_t gpuLocalId = d_buffers->gpus_g2l[gpuId];
    uint64_t slotId = d_buffers->streamsData[streamId].slotId;
    set_device(gpuId);

    SetupCtx *setupCtx = (SetupCtx *)pSetupCtx_;
    StepsParams *params = (StepsParams *)params_;
    hipStream_t stream = d_buffers->streamsData[streamId].stream;
    TimerGPU &timer = d_buffers->streamsData[streamId].timer;

    gl64_t *d_aux_trace = (gl64_t *)d_buffers->d_aux_trace[gpuLocalId] + slotId*d_buffers->max_size_prover_buffer;

    uint64_t N = (1 << setupCtx->starkInfo.starkStruct.nBits);
    uint64_t nCols = setupCtx->starkInfo.mapSectionsN["cm1"];
    uint64_t sizeTrace = N * (setupCtx->starkInfo.mapSectionsN["cm1"]) * sizeof(Goldilocks::Element);
    uint64_t sizeConstPols = N * (setupCtx->starkInfo.nConstants) * sizeof(Goldilocks::Element);
    uint64_t sizeConstTree = get_const_tree_size((void *)&setupCtx->starkInfo) * sizeof(Goldilocks::Element);
  
    auto key = std::make_pair(airgroupId, airId);
    std::string proofType = "basic";
    AirInstanceInfo *air_instance_info = d_buffers->air_instances[key][proofType][gpuLocalId];
    
    if (setupCtx->starkInfo.mapTotalNCustomCommitsFixed > 0) {
        Goldilocks::Element *pCustomCommitsFixed = (Goldilocks::Element *)d_aux_trace + setupCtx->starkInfo.mapOffsets[std::make_pair("custom_fixed", false)];
        CHECKCUDAERR(hipMemcpyAsync(pCustomCommitsFixed, params->pCustomCommitsFixed, setupCtx->starkInfo.mapTotalNCustomCommitsFixed * sizeof(Goldilocks::Element), hipMemcpyHostToDevice, stream));
    }

    d_buffers->streamsData[streamId].pSetupCtx = pSetupCtx_;
    d_buffers->streamsData[streamId].proofBuffer = proofBuffer;
    d_buffers->streamsData[streamId].proofFile = string(proofFile);
    d_buffers->streamsData[streamId].airgroupId = airgroupId;
    d_buffers->streamsData[streamId].airId = airId;
    d_buffers->streamsData[streamId].instanceId = instanceId;
    d_buffers->streamsData[streamId].proofType = "basic";

    uint64_t offsetStage1 = setupCtx->starkInfo.mapOffsets[std::make_pair("cm1", false)];
    uint64_t offsetPublicInputs = setupCtx->starkInfo.mapOffsets[std::make_pair("publics", false)];
    uint64_t offsetAirgroupValues = setupCtx->starkInfo.mapOffsets[std::make_pair("airgroupvalues", false)];
    uint64_t offsetAirValues = setupCtx->starkInfo.mapOffsets[std::make_pair("airvalues", false)];
    uint64_t offsetProofValues = setupCtx->starkInfo.mapOffsets[std::make_pair("proofvalues", false)];
    uint64_t offsetChallenge = setupCtx->starkInfo.mapOffsets[std::make_pair("challenge", false)];

    uint64_t blockSize = d_buffers->streamsData[streamId].pinned_size;
    Goldilocks::Element *pinned_buffer = d_buffers->streamsData[streamId].pinned_buffer;
    uint64_t copySize =0;
    uint64_t nBlocks = 0;

    if (!skipRecalculation) {
        copySize = N * nCols * sizeof(Goldilocks::Element);
        nBlocks = (copySize + blockSize - 1) / blockSize;
        for (uint64_t i = 0; i < nBlocks; ++i) {
            uint64_t copySizeBlock = std::min(blockSize, copySize - i * blockSize);
            memcpy(pinned_buffer, (uint8_t *)params->trace + i * blockSize, copySizeBlock);
            CHECKCUDAERR(hipMemcpyAsync((uint8_t*) (d_aux_trace + offsetStage1) + i*blockSize, pinned_buffer, copySizeBlock, hipMemcpyHostToDevice, stream));
            hipStreamSynchronize(stream);
        }        
    }
    
    copySize = setupCtx->starkInfo.nPublics * sizeof(Goldilocks::Element);
    nBlocks = (copySize + blockSize - 1) / blockSize;
    for (uint64_t i = 0; i < nBlocks; ++i) {
        uint64_t copySizeBlock = std::min(blockSize, copySize - i * blockSize);
        memcpy(pinned_buffer, (uint8_t *)params->publicInputs + i * blockSize, copySizeBlock);
        CHECKCUDAERR(hipMemcpyAsync((uint8_t*) (d_aux_trace + offsetPublicInputs) + i*blockSize, pinned_buffer, copySizeBlock, hipMemcpyHostToDevice, stream));
        hipStreamSynchronize(stream);
    }
    
    if (setupCtx->starkInfo.proofValuesSize > 0) {
        copySize = setupCtx->starkInfo.proofValuesSize * sizeof(Goldilocks::Element);
        nBlocks = (copySize + blockSize - 1) / blockSize;
        for (uint64_t i = 0; i < nBlocks; ++i) {
            uint64_t copySizeBlock = std::min(blockSize, copySize - i * blockSize);
            memcpy(pinned_buffer, (uint8_t *)params->proofValues + i * blockSize, copySizeBlock);
            CHECKCUDAERR(hipMemcpyAsync((uint8_t*) (d_aux_trace + offsetProofValues) + i*blockSize, pinned_buffer, copySizeBlock, hipMemcpyHostToDevice, stream));
            hipStreamSynchronize(stream);
        }
    }
    if (setupCtx->starkInfo.airgroupValuesSize > 0) {
        copySize = setupCtx->starkInfo.airgroupValuesSize * sizeof(Goldilocks::Element);
        nBlocks = (copySize + blockSize - 1) / blockSize;
        for (uint64_t i = 0; i < nBlocks; ++i) {
            uint64_t copySizeBlock = std::min(blockSize, copySize - i * blockSize);
            memcpy(pinned_buffer, (uint8_t *)params->airgroupValues + i * blockSize, copySizeBlock);
            CHECKCUDAERR(hipMemcpyAsync((uint8_t*) (d_aux_trace + offsetAirgroupValues) + i*blockSize, pinned_buffer, copySizeBlock, hipMemcpyHostToDevice, stream));
            hipStreamSynchronize(stream);
        }
    }
    if (setupCtx->starkInfo.airValuesSize > 0) {
        copySize = setupCtx->starkInfo.airValuesSize * sizeof(Goldilocks::Element);
        nBlocks = (copySize + blockSize - 1) / blockSize;
        for (uint64_t i = 0; i < nBlocks; ++i) {
            uint64_t copySizeBlock = std::min(blockSize, copySize - i * blockSize);
            memcpy(pinned_buffer, (uint8_t *)params->airValues + i * blockSize, copySizeBlock);
            CHECKCUDAERR(hipMemcpyAsync((uint8_t*) (d_aux_trace + offsetAirValues) + i*blockSize, pinned_buffer, copySizeBlock, hipMemcpyHostToDevice, stream));
            hipStreamSynchronize(stream);
        }
    }
    copySize = FIELD_EXTENSION * sizeof(Goldilocks::Element);
    memcpy(pinned_buffer, (uint8_t *)globalChallenge, copySize);
    CHECKCUDAERR(hipMemcpyAsync(d_aux_trace + offsetChallenge, pinned_buffer, copySize, hipMemcpyHostToDevice, stream));
    hipStreamSynchronize(stream);
    

    gl64_t *d_const_pols;
    gl64_t *d_const_tree;
    if (air_instance_info->stored) {
        d_const_pols = d_buffers->d_constPols[gpuLocalId] + air_instance_info->const_pols_offset;
        d_const_tree = d_buffers->d_constPols[gpuLocalId] + air_instance_info->const_tree_offset;
    } else {

        uint64_t offsetConstPols = setupCtx->starkInfo.mapOffsets[std::make_pair("const", false)];
        d_const_pols = d_aux_trace + offsetConstPols;
        uint32_t block_size = d_buffers->streamsData[streamId].pinned_size;
        uint32_t nBlocks = (sizeConstPols + block_size - 1) / block_size;
        Goldilocks::Element *pinned_buffer = d_buffers->streamsData[streamId].pinned_buffer;
        for(int i=0; i<nBlocks; ++i) {
            loadFileParallel_block(pinned_buffer, constPolsPath, block_size, true, i);
            uint64_t copy_size = std::min((uint64_t)block_size, sizeConstPols - (uint64_t)i * block_size);
            CHECKCUDAERR(hipMemcpyAsync((uint8_t*)d_const_pols + (uint64_t)i * block_size, pinned_buffer, copy_size, hipMemcpyHostToDevice, stream)); 
            hipStreamSynchronize(stream);
        }

        uint64_t offsetConstTree = setupCtx->starkInfo.mapOffsets[std::make_pair("const", true)];
        d_const_tree = d_aux_trace + offsetConstTree;

        nBlocks = (sizeConstTree + block_size - 1) / block_size;
        for(int i=0; i<nBlocks; ++i) {
            loadFileParallel_block(pinned_buffer, constTreePath, block_size, true, i);
            uint64_t copy_size_tree = std::min((uint64_t)block_size, sizeConstTree - (uint64_t)i * block_size);
            CHECKCUDAERR(hipMemcpyAsync((uint8_t*)d_const_tree + (uint64_t)i * block_size, pinned_buffer, copy_size_tree, hipMemcpyHostToDevice, stream)); 
            hipStreamSynchronize(stream);
        }

    }


    genProof_gpu(*setupCtx, d_aux_trace, d_const_pols, d_const_tree, streamId, instanceId, d_buffers, air_instance_info, skipRecalculation, timer, stream);
    hipEventRecord(d_buffers->streamsData[streamId].end_event, stream);
    d_buffers->streamsData[streamId].status = 2;
    return streamId;


}

void get_proof(DeviceCommitBuffers *d_buffers, uint64_t streamId) {

    uint32_t gpuId = d_buffers->streamsData[streamId].gpuId;
    set_device(gpuId);

    SetupCtx *setupCtx = (SetupCtx*) d_buffers->streamsData[streamId].pSetupCtx;
    uint64_t airgroupId = d_buffers->streamsData[streamId].airgroupId;
    uint64_t airId = d_buffers->streamsData[streamId].airId;
    uint64_t instanceId = d_buffers->streamsData[streamId].instanceId;
    uint64_t * proofBuffer = d_buffers->streamsData[streamId].proofBuffer;
    string proofType = d_buffers->streamsData[streamId].proofType;
    string proofFile = d_buffers->streamsData[streamId].proofFile;
    TimerGPU &timer = d_buffers->streamsData[streamId].timer;

    closeStreamTimer(timer, true);

    writeProof(*setupCtx, d_buffers->streamsData[streamId].pinned_buffer_proof, proofBuffer, airgroupId, airId, instanceId, proofFile);

    if (proof_done_callback != nullptr) {
        proof_done_callback(instanceId, proofType.c_str());
    }
}

void get_stream_proofs(void *d_buffers_){
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    std::lock_guard<std::mutex> lock(d_buffers->mutex_slot_selection);
    for (uint64_t i = 0; i < d_buffers->n_streams; i++) {
        if (d_buffers->streamsData[i].status == 0 || d_buffers->streamsData[i].status == 3) continue;
        set_device(d_buffers->streamsData[i].gpuId);
        CHECKCUDAERR(hipStreamSynchronize(d_buffers->streamsData[i].stream));
        if(d_buffers->streamsData[i].root != nullptr) {
            get_commit_root(d_buffers, i);
        }else{
            get_proof(d_buffers, i);
        }
        d_buffers->streamsData[i].reset();        
    }
}

void get_stream_proofs_non_blocking(void *d_buffers_){
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    std::lock_guard<std::mutex> lock(d_buffers->mutex_slot_selection);
    for (uint64_t i = 0; i < d_buffers->n_streams; i++) {
        if(d_buffers->streamsData[i].status==2 &&  hipEventQuery(d_buffers->streamsData[i].end_event) == hipSuccess){
            set_device(d_buffers->streamsData[i].gpuId);
            if(d_buffers->streamsData[i].root != nullptr) {
                get_commit_root(d_buffers, i);
            }else{
                get_proof(d_buffers, i);
            }
            d_buffers->streamsData[i].reset();        
        }
    }
}

void get_stream_id_proof(void *d_buffers_, uint64_t streamId) {
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    set_device(d_buffers->streamsData[streamId].gpuId);
    CHECKCUDAERR(hipStreamSynchronize(d_buffers->streamsData[streamId].stream));
    if(d_buffers->streamsData[streamId].root != nullptr) {
            get_commit_root(d_buffers, streamId);
        }else{
            get_proof(d_buffers, streamId);
        }
    d_buffers->streamsData[streamId].reset();
}

uint64_t gen_recursive_proof(void *pSetupCtx_, char *globalInfoFile, uint64_t airgroupId, uint64_t airId, uint64_t instanceId, void *trace, void *aux_trace, void *pConstPols, void *pConstTree, void *pPublicInputs, uint64_t* proofBuffer, char *proof_file, bool vadcop, void *d_buffers_, char *constPolsPath, char *constTreePath, char *proofType)
{
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    uint32_t streamId = selectStream(d_buffers);
    uint32_t gpuId = d_buffers->streamsData[streamId].gpuId;
    uint32_t gpuLocalId = d_buffers->gpus_g2l[gpuId];
    uint64_t slotId =  d_buffers->streamsData[streamId].slotId;
    set_device(gpuId);

    SetupCtx *setupCtx = (SetupCtx *)pSetupCtx_;
    hipStream_t stream = d_buffers->streamsData[streamId].stream;
    TimerGPU &timer = d_buffers->streamsData[streamId].timer;
    
    gl64_t *d_trace = (gl64_t *)d_buffers->d_aux_trace[gpuLocalId] + slotId*d_buffers->max_size_prover_buffer;
    gl64_t *d_aux_trace = d_trace + d_buffers->max_size_trace_aggregation;

    uint64_t N = (1 << setupCtx->starkInfo.starkStruct.nBits);
    uint64_t nCols = setupCtx->starkInfo.mapSectionsN["cm1"];
    uint64_t sizeTrace = N * nCols * sizeof(Goldilocks::Element);
    uint64_t sizeConstPols = N * (setupCtx->starkInfo.nConstants) * sizeof(Goldilocks::Element);
    uint64_t sizeConstTree = get_const_tree_size((void *)&setupCtx->starkInfo) * sizeof(Goldilocks::Element);

    auto key = std::make_pair(airgroupId, airId);
    AirInstanceInfo *air_instance_info = d_buffers->air_instances[key][string(proofType)][gpuLocalId];


    d_buffers->streamsData[streamId].pSetupCtx = pSetupCtx_;
    d_buffers->streamsData[streamId].proofBuffer = proofBuffer;
    d_buffers->streamsData[streamId].proofFile = string(proof_file);
    d_buffers->streamsData[streamId].airgroupId = airgroupId;
    d_buffers->streamsData[streamId].airId = airId;
    d_buffers->streamsData[streamId].instanceId = instanceId;
    d_buffers->streamsData[streamId].proofType = string(proofType);

    Goldilocks::Element *pinned_buffer = d_buffers->streamsData[streamId].pinned_buffer;
    uint64_t blockSize = d_buffers->streamsData[streamId].pinned_size;
    uint64_t copySize = sizeTrace;
    uint64_t nBlocks = (copySize + blockSize - 1) / blockSize;
    for (uint64_t i = 0; i < nBlocks; ++i) {
        uint64_t copySizeBlock = std::min(blockSize, copySize - i * blockSize);
        memcpy(pinned_buffer, (uint8_t *)trace + i * blockSize, copySizeBlock);
        CHECKCUDAERR(hipMemcpyAsync((uint8_t*)d_trace + i*blockSize, pinned_buffer, copySizeBlock, hipMemcpyHostToDevice, stream));
        hipStreamSynchronize(stream);
    }
    uint64_t offsetPublicInputs = setupCtx->starkInfo.mapOffsets[std::make_pair("publics", false)];
    copySize = setupCtx->starkInfo.nPublics * sizeof(Goldilocks::Element);
    nBlocks = (copySize + blockSize - 1) / blockSize;
    for (uint64_t i = 0; i < nBlocks; ++i) {
        uint64_t copySizeBlock = std::min(blockSize, copySize - i * blockSize);
        memcpy(pinned_buffer, (uint8_t *)pPublicInputs + i * blockSize, copySizeBlock);
        CHECKCUDAERR(hipMemcpyAsync((uint8_t*)(d_aux_trace + offsetPublicInputs) + i*blockSize, pinned_buffer, copySizeBlock, hipMemcpyHostToDevice, stream));
        hipStreamSynchronize(stream);
    }
    
    gl64_t *d_const_pols;
    gl64_t *d_const_tree;
    if (air_instance_info->stored) {
        d_const_pols = d_buffers->d_constPolsAggregation[gpuLocalId] + air_instance_info->const_pols_offset;
        d_const_tree = d_buffers->d_constPolsAggregation[gpuLocalId] + air_instance_info->const_tree_offset;
    } else {
        uint64_t offsetConstPols = setupCtx->starkInfo.mapOffsets[std::make_pair("const", false)];
        d_const_pols = d_aux_trace + offsetConstPols;
        uint32_t block_size = d_buffers->streamsData[streamId].pinned_size;
        uint32_t nBlocks = (sizeConstPols + block_size - 1) / block_size;
        Goldilocks::Element *pinned_buffer = d_buffers->streamsData[streamId].pinned_buffer;
        for(int i=0; i<nBlocks; ++i) {
            loadFileParallel_block(pinned_buffer, constPolsPath, block_size, true, i);
            uint64_t copy_size = std::min((uint64_t)block_size, sizeConstPols - (uint64_t)i * block_size);
            CHECKCUDAERR(hipMemcpyAsync((uint8_t*)d_const_pols + (uint64_t)i * block_size, pinned_buffer, copy_size, hipMemcpyHostToDevice, stream)); 
            hipStreamSynchronize(stream);
        }

        uint64_t offsetConstTree = setupCtx->starkInfo.mapOffsets[std::make_pair("const", true)];
        d_const_tree = d_aux_trace + offsetConstTree;
        nBlocks = (sizeConstTree + block_size - 1) / block_size;
        for(int i=0; i<nBlocks; ++i) {
            loadFileParallel_block(pinned_buffer, constTreePath, block_size, true, i);
            uint64_t copy_size_tree = std::min((uint64_t)block_size, sizeConstTree - (uint64_t)i * block_size);
            CHECKCUDAERR(hipMemcpyAsync((uint8_t*)d_const_tree + (uint64_t)i * block_size, pinned_buffer, copy_size_tree, hipMemcpyHostToDevice, stream)); 
            hipStreamSynchronize(stream);
        }
    }

    genRecursiveProof_gpu<Goldilocks::Element>(*setupCtx, d_trace, d_aux_trace, d_const_pols, d_const_tree, streamId, d_buffers, air_instance_info, instanceId, timer, stream);
    hipEventRecord(d_buffers->streamsData[streamId].end_event, stream);
    d_buffers->streamsData[streamId].status = 2;
    return streamId;
}

uint64_t commit_witness(uint64_t arity, uint64_t nBits, uint64_t nBitsExt, uint64_t nCols, uint64_t instanceId, void *root, void *trace, void *auxTrace, void *d_buffers_, void *pSetupCtx_) {

    SetupCtx *setupCtx = (SetupCtx *)pSetupCtx_;
    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    uint32_t streamId = selectStream(d_buffers);
    uint32_t gpuId = d_buffers->streamsData[streamId].gpuId;
    uint32_t gpuLocalId = d_buffers->gpus_g2l[gpuId];
    uint64_t slotId = d_buffers->streamsData[streamId].slotId;
    set_device(gpuId);

    d_buffers->streamsData[streamId].root = root;
    d_buffers->streamsData[streamId].instanceId = instanceId;

    uint64_t N = 1 << nBits;

    hipStream_t stream = d_buffers->streamsData[streamId].stream;
    TimerGPU &timer = d_buffers->streamsData[streamId].timer;

    gl64_t *d_aux_trace = (gl64_t *)d_buffers->d_aux_trace[gpuLocalId] + slotId*d_buffers->max_size_prover_buffer;
    uint64_t sizeTrace = N * nCols * sizeof(Goldilocks::Element);
    uint64_t offsetStage1 = setupCtx->starkInfo.mapOffsets[std::make_pair("cm1", false)];

    uint64_t blockSize = d_buffers->streamsData[streamId].pinned_size;
    uint64_t copySize = sizeTrace;
    uint64_t nBlocks = (copySize + blockSize - 1) / blockSize;
    Goldilocks::Element *pinned_buffer = d_buffers->streamsData[streamId].pinned_buffer;
    for (uint64_t i = 0; i < nBlocks; ++i) {
        uint64_t copySizeBlock = std::min(blockSize, copySize - i * blockSize);
        memcpy(pinned_buffer, (uint8_t *)trace + i * blockSize, copySizeBlock);
        CHECKCUDAERR(hipMemcpyAsync((uint8_t*)d_aux_trace + offsetStage1 + i*blockSize, pinned_buffer, copySizeBlock, hipMemcpyHostToDevice, stream));
        hipStreamSynchronize(stream);
    }
    genCommit_gpu(arity, nBits, nBitsExt, nCols, d_aux_trace, d_buffers->streamsData[streamId].pinned_buffer_proof, setupCtx, timer, stream);

    hipEventRecord(d_buffers->streamsData[streamId].end_event, stream);
    d_buffers->streamsData[streamId].status = 2;
    return streamId;
}

void get_commit_root(DeviceCommitBuffers *d_buffers, uint64_t streamId) {

    set_device(d_buffers->streamsData[streamId].gpuId);
    Goldilocks::Element *root = (Goldilocks::Element *)d_buffers->streamsData[streamId].root;
    memcpy((Goldilocks::Element *)root, d_buffers->streamsData[streamId].pinned_buffer_proof, HASH_SIZE * sizeof(uint64_t));
    closeStreamTimer(d_buffers->streamsData[streamId].timer, false);
    
    uint64_t instanceId = d_buffers->streamsData[streamId].instanceId;

    if (proof_done_callback != nullptr) {
        proof_done_callback(instanceId, "");
    }

}

uint64_t check_device_memory() {
    
    set_device(0); //We assume that all the GPUs have the same characteristics, we only check the GPU 0
    uint64_t freeMem, totalMem;
    hipError_t err = hipMemGetInfo(&freeMem, &totalMem);
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return 0;
    }

    zklog.trace("Free memory GPU: " +  to_string(freeMem / (1024.0 * 1024.0)) + " MB");
    zklog.trace("Total memory GPU: " + to_string(totalMem / (1024.0 * 1024.0)) + " MB");

    return freeMem;
}

uint64_t get_num_gpus() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    return deviceCount;
}

// Function to set the CUDA device based on the MPI rank
void set_device_mpi(uint32_t mpi_node_rank){
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found." << std::endl;
        exit(1);
    }
    int device = mpi_node_rank % deviceCount;
    hipSetDevice(device);
}

void set_device(uint32_t gpuId){
    hipSetDevice(gpuId);
}

uint32_t selectStream(DeviceCommitBuffers* d_buffers){

    std::lock_guard<std::mutex> lock(d_buffers->mutex_slot_selection);
    uint32_t countFreeStreamsGPU[d_buffers->n_gpus];
    uint32_t countUnusedStreams[d_buffers->n_gpus];
    int streamIdxGPU[d_buffers->n_gpus];
    
    for( uint32_t i = 0; i < d_buffers->n_gpus; i++){
        countUnusedStreams[i] = 0;
        countFreeStreamsGPU[i] = 0;
        streamIdxGPU[i] = -1;
    }

    bool someFree = false;
    while (!someFree){
        for (uint32_t i = 0; i < d_buffers->n_streams; i++) {
            if (d_buffers->streamsData[i].status==0 || d_buffers->streamsData[i].status==3 || (d_buffers->streamsData[i].status==2 &&  hipEventQuery(d_buffers->streamsData[i].end_event) == hipSuccess)) {

                countFreeStreamsGPU[d_buffers->gpus_g2l[d_buffers->streamsData[i].gpuId]]++;
                if(d_buffers->streamsData[i].status==0){
                    countUnusedStreams[d_buffers->gpus_g2l[d_buffers->streamsData[i].gpuId]]++;
                    streamIdxGPU[d_buffers->gpus_g2l[d_buffers->streamsData[i].gpuId]] = i;
                }
                if( streamIdxGPU[d_buffers->gpus_g2l[d_buffers->streamsData[i].gpuId]] == -1 ){
                    streamIdxGPU[d_buffers->gpus_g2l[d_buffers->streamsData[i].gpuId]] = i;
                }
                someFree = true;
            }
        }
        if (!someFree)
            std::this_thread::sleep_for(std::chrono::microseconds(300)); 
    }

    uint32_t maxFree = 0;
    uint32_t streamId = 0;
    for (uint32_t i = 0; i < d_buffers->n_gpus; i++) {
        if (countFreeStreamsGPU[i] > maxFree || (countFreeStreamsGPU[i] == maxFree && countUnusedStreams[i] > countUnusedStreams[streamId])) {
            maxFree = countFreeStreamsGPU[i];
            streamId = streamIdxGPU[i];
        }
    }

    reserveStream(d_buffers, streamId);
    return streamId;
}

void reserveStream(DeviceCommitBuffers* d_buffers, uint32_t streamId){
    if(d_buffers->streamsData[streamId].status==2 &&  hipEventQuery(d_buffers->streamsData[streamId].end_event) == hipSuccess) {

        if(d_buffers->streamsData[streamId].root != nullptr) {
            get_commit_root(d_buffers, streamId);
        }else{
            get_proof(d_buffers, streamId);
        }
        d_buffers->streamsData[streamId].reset();
    }

    d_buffers->streamsData[streamId].status = 1;
}

void closeStreamTimer(TimerGPU &timer, bool isProve) {
    TimerSyncAndLogAllGPU(timer); 
    TimerSyncCategoriesGPU(timer);
    if(isProve)
        TimerLogCategoryContributionsGPU(timer, STARK_GPU_PROOF);
    else
        TimerLogCategoryContributionsGPU(timer, STARK_GPU_COMMIT);
    TimerResetGPU(timer);
}
#endif