#include "zkglobals.hpp"
#include "proof2zkinStark.hpp"
#include "starks.hpp"
#include "omp.h"
#include "starks_api.hpp"
#ifdef __USE_CUDA__
#include "gen_recursive_proof.cuh"
#include "gen_proof.cuh"

struct MaxSizes
{
    uint64_t maxN;
    uint64_t maxNExtended;
    uint64_t maxTraceArea;
    uint64_t maxNTTArea;
    uint64_t maxConstArea;
    uint64_t maxNPublics;
    uint64_t maxAuxTraceArea;
    uint64_t maxConstTreeSize;
};


void *gen_device_commit_buffers(void *maxSizes_)
{
    MaxSizes *maxSizes = (MaxSizes *)maxSizes_;
    CHECKCUDAERR(hipSetDevice(0));
    DeviceCommitBuffers *buffers = new DeviceCommitBuffers();
    CHECKCUDAERR(hipMalloc(&buffers->d_trace, maxSizes->maxTraceArea * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_constPols, maxSizes->maxConstArea * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_constTree, maxSizes->maxConstTreeSize * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_publicInputs, maxSizes->maxNPublics * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_aux_trace, maxSizes->maxAuxTraceArea * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_forwardTwiddleFactors, maxSizes->maxNExtended * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_inverseTwiddleFactors, maxSizes->maxNExtended * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_r, maxSizes->maxNExtended * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_ntt, maxSizes->maxNTTArea * sizeof(Goldilocks::Element)));
    CHECKCUDAERR(hipMalloc(&buffers->d_tree, maxSizes->maxNExtended * sizeof(uint64_t)));
    return (void *)buffers;

}
void gen_proof(void *pSetupCtx, uint64_t airgroupId, uint64_t airId, uint64_t instanceId, void *params, void *globalChallenge, void* pBuffHelper, uint64_t* proofBuffer, char *proofFile) {

    SetupCtx *setupCtx = (SetupCtx *)pSetupCtx;

    genProof_gpu(*(SetupCtx *)pSetupCtx, airgroupId, airId, instanceId, *(StepsParams *)params, (Goldilocks::Element *)globalChallenge, (Goldilocks::Element *)pBuffHelper, proofBuffer, string(proofFile));
}

void gen_recursive_proof(void *pSetupCtx_, char *globalInfoFile, uint64_t airgroupId, uint64_t airId, uint64_t instanceId, void *trace, void *aux_trace, void *pConstPols, void *pConstTree, void *pPublicInputs, uint64_t* proofBuffer, char *proof_file, bool vadcop, void *d_buffers_)
{

    json globalInfo;
    file2json(globalInfoFile, globalInfo);

    DeviceCommitBuffers *d_buffers = (DeviceCommitBuffers *)d_buffers_;
    SetupCtx *setupCtx = (SetupCtx *)pSetupCtx_;
    double time = omp_get_wtime();

    CHECKCUDAERR(hipSetDevice(0));
    uint64_t N = (1 << setupCtx->starkInfo.starkStruct.nBits);
    uint64_t sizeWitness = N * (setupCtx->starkInfo.mapSectionsN["cm1"]) * sizeof(Goldilocks::Element);
    uint64_t sizeConstPols = N * (setupCtx->starkInfo.nConstants) * sizeof(Goldilocks::Element);
    uint64_t sizeConstTree = get_const_tree_size((void *)&setupCtx->starkInfo);

    CHECKCUDAERR(hipMemcpy(d_buffers->d_trace, trace, sizeWitness, hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(d_buffers->d_constPols, pConstPols, sizeConstPols, hipMemcpyHostToDevice));
    CHECKCUDAERR(hipMemcpy(d_buffers->d_constTree, pConstTree, sizeConstTree*sizeof(Goldilocks::Element), hipMemcpyHostToDevice));

    time = omp_get_wtime() - time;
    std::cout << "rick genDeviceBuffers time: " << time << std::endl;

    time = omp_get_wtime();
    genRecursiveProof_gpu<Goldilocks::Element>(*setupCtx, globalInfo, airgroupId, airId, instanceId, (Goldilocks::Element *)trace, (Goldilocks::Element *)pConstPols, (Goldilocks::Element *)pConstTree, (Goldilocks::Element *)pPublicInputs, proofBuffer, string(proof_file), d_buffers, vadcop);
    time = omp_get_wtime() - time;
    std::cout << "rick genRecursiveProof_gpu time: " << time << std::endl;
}
#endif