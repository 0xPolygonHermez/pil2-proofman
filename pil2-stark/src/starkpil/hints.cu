#include "hints.hpp"
#include "expressions_gpu.cuh"

void opHintFieldsGPU(SetupCtx& setupCtx, StepsParams& params, StepsParams& d_params, std::vector<Dest> &dests, void* GPUExpressionsCtx){

    ExpressionsGPU* expressionsCtx = (ExpressionsGPU*)GPUExpressionsCtx;
    uint64_t domainSize = 1 << setupCtx.starkInfo.starkStruct.nBits;
    expressionsCtx->calculateExpressions_gpu(params, d_params, setupCtx.expressionsBin.expressionsBinArgsExpressions, dests, domainSize);
}

void allocateDestGPU(Goldilocks::Element**buff, uint64_t size){
    hipMalloc((void**) buff, size * sizeof(Goldilocks::Element));
}
void freeDestGPU(Goldilocks::Element* buff){
    hipFree(buff);
}

void setPolynomialGPU(SetupCtx& setupCtx, Goldilocks::Element *buffer, Goldilocks::Element *values, uint64_t idPol) {
    /*PolMap polInfo = setupCtx.starkInfo.cmPolsMap[idPol];
    uint64_t deg = 1 << setupCtx.starkInfo.starkStruct.nBits;
    uint64_t dim = polInfo.dim;
    std::string stage = "cm" + to_string(polInfo.stage);
    uint64_t nCols = setupCtx.starkInfo.mapSectionsN[stage];
    uint64_t offset = setupCtx.starkInfo.mapOffsets[std::make_pair(stage, false)];
    offset += polInfo.stagePos;
#pragma omp parallel for
    for(uint64_t j = 0; j < deg; ++j) {
        CHECKCUDAERR(hipMemcpy(buffer + offset + j * nCols, &values[j * dim], dim * sizeof(Goldilocks::Element), hipMemcpyHostToDevice));
    }*/

    PolMap polInfo = setupCtx.starkInfo.cmPolsMap[idPol];
    uint64_t deg = 1 << setupCtx.starkInfo.starkStruct.nBits;
    uint64_t dim = polInfo.dim;
    std::string stage = "cm" + to_string(polInfo.stage);
    uint64_t nCols = setupCtx.starkInfo.mapSectionsN[stage];
    Goldilocks::Element* auxSection = new Goldilocks::Element[nCols * deg];
    uint64_t sectionOffest = setupCtx.starkInfo.mapOffsets[std::make_pair(stage, false)];
    CHECKCUDAERR(hipMemcpy(auxSection, buffer + sectionOffest, nCols * deg * sizeof(Goldilocks::Element), hipMemcpyDeviceToHost));
    uint64_t offset = polInfo.stagePos;
    Polinomial pol = Polinomial(&auxSection[offset], deg, dim, nCols, std::to_string(idPol));
#pragma omp parallel for
    for(uint64_t j = 0; j < deg; ++j) {
        std::memcpy(pol[j], &values[j*dim], dim * sizeof(Goldilocks::Element));
    }
    CHECKCUDAERR(hipMemcpy(buffer + sectionOffest, auxSection, nCols * deg * sizeof(Goldilocks::Element), hipMemcpyHostToDevice));
    delete[] auxSection;
}
