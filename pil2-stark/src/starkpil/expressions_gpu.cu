#include "hip/hip_runtime.h"
#include "expressions_gpu.cuh"
#include "cuda_utils.cuh"
#include "cuda_utils.hpp"
#include "gl64_t.cuh"
#include "goldilocks_cubic_extension.cuh"

ExpressionsGPU::ExpressionsGPU(SetupCtx &setupCtx, uint32_t nParamsMax, uint32_t nTemp1Max, uint32_t nTemp3Max, uint64_t nrowsPack_, uint32_t nBlocks_) : ExpressionsCtx(setupCtx), nParamsMax(nParamsMax), nTemp1Max(nTemp1Max), nTemp3Max(nTemp3Max), nrowsPack(nrowsPack_), nBlocks(nBlocks_)
{
    uint64_t nOpenings = setupCtx.starkInfo.openingPoints.size();
    uint64_t ns = 2 + setupCtx.starkInfo.nStages + setupCtx.starkInfo.customCommits.size();
    vector<uint64_t> nColsStages(ns * nOpenings + 1, 0);
    vector<uint64_t> nColsStagesAcc(ns * nOpenings + 1, 0);

    for (uint64_t o = 0; o < nOpenings; ++o)
    {
        for (uint64_t stage = 0; stage < ns; ++stage)
        {
            if (stage == 0)
            {
                nColsStages[ns * o] = setupCtx.starkInfo.mapSectionsN["const"];
                nColsStagesAcc[ns * o] = o == 0 ? 0 : nColsStagesAcc[ns * o + stage - 1] + nColsStages[stage - 1];
            }
            else if (stage < 2 + setupCtx.starkInfo.nStages)
            {
                std::string section = "cm" + to_string(stage);
                nColsStages[ns * o + stage] = setupCtx.starkInfo.mapSectionsN[section];
                nColsStagesAcc[ns * o + stage] = nColsStagesAcc[ns * o + stage - 1] + nColsStages[stage - 1];
            }
            else
            {
                uint64_t index = stage - setupCtx.starkInfo.nStages - 2;
                std::string section = setupCtx.starkInfo.customCommits[index].name + "0";
                nColsStages[ns * o + stage] = setupCtx.starkInfo.mapSectionsN[section];
                nColsStagesAcc[ns * o + stage] = nColsStagesAcc[ns * o + stage - 1] + nColsStages[stage - 1];
            }
        }
    }
    nColsStagesAcc[ns * nOpenings] = nColsStagesAcc[ns * nOpenings - 1] + nColsStages[ns * nOpenings - 1];

    uint64_t Nexteded = 1 << setupCtx.starkInfo.starkStruct.nBitsExt;
    uint64_t N = 1 << setupCtx.starkInfo.starkStruct.nBits;
    hipMalloc(&h_deviceArgs.nextStrides, nOpenings * sizeof(uint64_t));
    hipMalloc(&h_deviceArgs.nColsStages, (ns * nOpenings + 1) * sizeof(uint64_t));
    hipMalloc(&h_deviceArgs.nColsStagesAcc, (ns * nOpenings + 1) * sizeof(uint64_t));
    hipMalloc(&h_deviceArgs.offsetsStages, (ns * nOpenings + 1) * sizeof(uint64_t));
    hipMalloc(&h_deviceArgs.cmPolsInfo, setupCtx.starkInfo.cmPolsMap.size() * 3 * sizeof(uint64_t));
    hipMalloc(&h_deviceArgs.zi, setupCtx.starkInfo.boundaries.size() * Nexteded * sizeof(Goldilocks::Element)); // rick: pillar calculada abans
    hipMalloc(&h_deviceArgs.x_n, N * sizeof(Goldilocks::Element));
    hipMalloc(&h_deviceArgs.x_2ns, Nexteded * sizeof(Goldilocks::Element));
    hipMalloc(&h_deviceArgs.challenges, setupCtx.starkInfo.challengesMap.size() * FIELD_EXTENSION * sizeof(Goldilocks::Element));
    uint64_t nNumbers = setupCtx.expressionsBin.expressionsBinArgsExpressions.nNumbers;
    hipMalloc(&h_deviceArgs.numbers, nNumbers * sizeof(Goldilocks::Element));
    hipMalloc(&h_deviceArgs.publics, setupCtx.starkInfo.nPublics * sizeof(Goldilocks::Element));
    hipMalloc(&h_deviceArgs.evals, setupCtx.starkInfo.evMap.size() * FIELD_EXTENSION * sizeof(Goldilocks::Element));
    hipMalloc(&h_deviceArgs.airgroupValues, setupCtx.starkInfo.airgroupValuesMap.size() * FIELD_EXTENSION * sizeof(Goldilocks::Element));
    hipMalloc(&h_deviceArgs.airValues, setupCtx.starkInfo.airValuesMap.size() * FIELD_EXTENSION * sizeof(Goldilocks::Element));
    hipMalloc(&h_deviceArgs.proofValues, setupCtx.starkInfo.proofValuesMap.size() * FIELD_EXTENSION * sizeof(Goldilocks::Element));
    hipMalloc(&h_deviceArgs.ops, setupCtx.expressionsBin.expressionsBinArgsExpressions.nOpsTotal * sizeof(uint8_t));
    hipMalloc(&h_deviceArgs.args, setupCtx.expressionsBin.expressionsBinArgsExpressions.nArgsTotal * sizeof(uint16_t));

    uint64_t nCols1 = nColsStagesAcc[ns * nOpenings] + setupCtx.starkInfo.boundaries.size() + 1;
    uint64_t nCols2 = nColsStagesAcc[ns * nOpenings] + nOpenings * FIELD_EXTENSION;
    uint64_t nCols3 = nColsStagesAcc[ns * nOpenings] + 1;
    uint64_t nColsMax = max(nCols1, max(nCols2, nCols3));

    // bufferT_
    h_deviceArgs.bufferSize = nOpenings * nrowsPack * nColsMax; // this must be moved from here

    while(nBlocks * h_deviceArgs.bufferSize * sizeof(Goldilocks::Element) > 1024 * 1024 * 1024 * 16){
        nBlocks = nBlocks / 2;
        nrowsPack = nrowsPack / 2;
        h_deviceArgs.bufferSize = nOpenings * nrowsPack * nColsMax;
    }
    hipMalloc(&h_deviceArgs.bufferT_, nBlocks * h_deviceArgs.bufferSize * sizeof(Goldilocks::Element));
    std::cout << "Total memory in expressions buffers [Gb]: " << (1.0 * nBlocks * h_deviceArgs.bufferSize * sizeof(Goldilocks::Element)) / (1024.0 * 1024.0 * 1024.0) << std::endl;

    // destVals
    h_deviceArgs.destValsSize = nParamsMax * FIELD_EXTENSION * nrowsPack;
    hipMalloc(&h_deviceArgs.destVals, nBlocks * h_deviceArgs.destValsSize * sizeof(Goldilocks::Element));
    std::cout << "Total memory in expressions destVals [Gb]: " << (1.0 * nBlocks * h_deviceArgs.destValsSize * sizeof(Goldilocks::Element)) / (1024.0 * 1024.0 * 1024.0) << std::endl;

    // tmps
    h_deviceArgs.tmp1Size = nTemp1Max * nrowsPack;
    h_deviceArgs.tmp3Size = nTemp3Max * FIELD_EXTENSION * nrowsPack;
    hipMalloc(&h_deviceArgs.tmp1, nBlocks * h_deviceArgs.tmp1Size * sizeof(Goldilocks::Element));
    hipMalloc(&h_deviceArgs.tmp3, nBlocks * h_deviceArgs.tmp3Size * sizeof(Goldilocks::Element));
    std::cout << "Total memory in expressions tmp1 [Gb]: " << (1.0 * nBlocks * h_deviceArgs.tmp1Size * sizeof(Goldilocks::Element)) / (1024.0 * 1024.0 * 1024.0) << std::endl;
    std::cout << "Total memory in expressions tmp3 [Gb]: " << (1.0 * nBlocks * h_deviceArgs.tmp3Size * sizeof(Goldilocks::Element)) / (1024.0 * 1024.0 * 1024.0) << std::endl;

    // constant deviceArgs
    hipMemcpy(h_deviceArgs.nColsStages, nColsStages.data(), nColsStages.size() * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(h_deviceArgs.nColsStagesAcc, nColsStagesAcc.data(), nColsStagesAcc.size() * sizeof(uint64_t), hipMemcpyHostToDevice);
    h_deviceArgs.N = 1 << setupCtx.starkInfo.starkStruct.nBits;
    h_deviceArgs.NExtended = 1 << setupCtx.starkInfo.starkStruct.nBitsExt;
    h_deviceArgs.nrowsPack = nrowsPack;
    h_deviceArgs.nOpenings = nOpenings;
    h_deviceArgs.ns = ns;

    // Size fixed customCommit
    if (setupCtx.starkInfo.mapTotalNCustomCommitsFixed > 0)
    {
        h_deviceArgs.customCommitsCols = setupCtx.starkInfo.customCommits[0].stageWidths[0];
    }
    else
    {
        h_deviceArgs.customCommitsCols = 0;
    }
};

ExpressionsGPU::~ExpressionsGPU()
{
    hipFree(h_deviceArgs.nextStrides);
    hipFree(h_deviceArgs.nColsStages);
    hipFree(h_deviceArgs.nColsStagesAcc);
    hipFree(h_deviceArgs.offsetsStages);
    hipFree(h_deviceArgs.cmPolsInfo);
    hipFree(h_deviceArgs.zi);
    hipFree(h_deviceArgs.x_n);
    hipFree(h_deviceArgs.x_2ns);
    hipFree(h_deviceArgs.challenges);
    hipFree(h_deviceArgs.numbers);
    hipFree(h_deviceArgs.publics);
    hipFree(h_deviceArgs.evals);
    hipFree(h_deviceArgs.airgroupValues);
    hipFree(h_deviceArgs.airValues);
    hipFree(h_deviceArgs.proofValues);
    hipFree(h_deviceArgs.ops);
    hipFree(h_deviceArgs.args);
    hipFree(h_deviceArgs.bufferT_);
    hipFree(h_deviceArgs.destVals);
    hipFree(h_deviceArgs.tmp1);
    hipFree(h_deviceArgs.tmp3);
}

void ExpressionsGPU::setBufferTInfo(uint64_t domainSize, StepsParams &params, StepsParams &params_gpu, ParserArgs &parserArgs, std::vector<Dest> &dests)
{

    bool domainExtended = domainSize == uint64_t(1 << setupCtx.starkInfo.starkStruct.nBitsExt) ? true : false;
    uint64_t expId = dests[0].params[0].op == opType::tmp ? dests[0].params[0].parserParams.destDim : 0;
    uint64_t nOpenings = h_deviceArgs.nOpenings;
    uint64_t ns = h_deviceArgs.ns;
    vector<uint64_t> nColsStages(ns * nOpenings + 1, 0);
    vector<uint64_t> nColsStagesAcc(ns * nOpenings + 1, 0);
    vector<uint64_t> offsetsStages(ns * nOpenings + 1, 0);

    for (uint64_t o = 0; o < nOpenings; ++o)
    {
        for (uint64_t stage = 0; stage < ns; ++stage)
        {
            if (stage == 0)
            {
                offsetsStages[ns * o] = 0;
                nColsStages[ns * o] = setupCtx.starkInfo.mapSectionsN["const"];
                nColsStagesAcc[ns * o] = o == 0 ? 0 : nColsStagesAcc[ns * o + stage - 1] + nColsStages[stage - 1];
            }
            else if (stage < 2 + setupCtx.starkInfo.nStages)
            {
                std::string section = "cm" + to_string(stage);
                offsetsStages[ns * o + stage] = setupCtx.starkInfo.mapOffsets[std::make_pair(section, domainExtended)];
                nColsStages[ns * o + stage] = setupCtx.starkInfo.mapSectionsN[section];
                nColsStagesAcc[ns * o + stage] = nColsStagesAcc[ns * o + stage - 1] + nColsStages[stage - 1];
            }
            else
            {
                uint64_t index = stage - setupCtx.starkInfo.nStages - 2;
                std::string section = setupCtx.starkInfo.customCommits[index].name + "0";
                offsetsStages[ns * o + stage] = setupCtx.starkInfo.mapOffsets[std::make_pair(section, domainExtended)];
                nColsStages[ns * o + stage] = setupCtx.starkInfo.mapSectionsN[section];
                nColsStagesAcc[ns * o + stage] = nColsStagesAcc[ns * o + stage - 1] + nColsStages[stage - 1];
            }
        }
    }

    nColsStagesAcc[ns * nOpenings] = nColsStagesAcc[ns * nOpenings - 1] + nColsStages[ns * nOpenings - 1];
    if (expId == int64_t(setupCtx.starkInfo.cExpId))
    {
        nCols = nColsStagesAcc[ns * nOpenings] + setupCtx.starkInfo.boundaries.size() + 1;
    }
    else if (expId == int64_t(setupCtx.starkInfo.friExpId))
    {
        nCols = nColsStagesAcc[ns * nOpenings] + nOpenings * FIELD_EXTENSION;
    }
    else
    {
        nCols = nColsStagesAcc[ns * nOpenings] + 1;
    }

    // fill device arguments
    h_deviceArgs.domainSize = domainSize;
    h_deviceArgs.nCols = nCols;
    h_deviceArgs.domainExtended = domainExtended;

    uint32_t extendBits = (setupCtx.starkInfo.starkStruct.nBitsExt - setupCtx.starkInfo.starkStruct.nBits);
    int64_t extend = domainExtended ? (1 << extendBits) : 1;
    uint64_t *nextStrides = new uint64_t[nOpenings];
    for (uint64_t i = 0; i < nOpenings; ++i)
    {
        uint64_t opening = setupCtx.starkInfo.openingPoints[i] < 0 ? setupCtx.starkInfo.openingPoints[i] + domainSize : setupCtx.starkInfo.openingPoints[i];
        nextStrides[i] = opening * extend;
    }
    hipMemcpy(h_deviceArgs.nextStrides, nextStrides, h_deviceArgs.nOpenings * sizeof(uint64_t), hipMemcpyHostToDevice);
    delete[] nextStrides;
    CHECKCUDAERR(hipGetLastError());


    hipMemcpy(h_deviceArgs.offsetsStages, offsetsStages.data(), offsetsStages.size() * sizeof(uint64_t), hipMemcpyHostToDevice);
    CHECKCUDAERR(hipGetLastError());

    h_deviceArgs.constPolsSize = setupCtx.starkInfo.nConstants;
    h_deviceArgs.cmPolsInfoSize = setupCtx.starkInfo.cmPolsMap.size();
    uint64_t *cmPolsInfo = new uint64_t[h_deviceArgs.cmPolsInfoSize * 3];
    for (uint64_t i = 0; i < h_deviceArgs.cmPolsInfoSize; ++i)
    {
        cmPolsInfo[i * 3] = setupCtx.starkInfo.cmPolsMap[i].stage;
        cmPolsInfo[i * 3 + 1] = setupCtx.starkInfo.cmPolsMap[i].stagePos;
        cmPolsInfo[i * 3 + 2] = setupCtx.starkInfo.cmPolsMap[i].dim;
    }
    hipMemcpy(h_deviceArgs.cmPolsInfo, cmPolsInfo, h_deviceArgs.cmPolsInfoSize * 3 * sizeof(uint64_t), hipMemcpyHostToDevice);
    CHECKCUDAERR(hipGetLastError());

    free(cmPolsInfo);
    if (dests[0].params[0].parserParams.expId == int64_t(setupCtx.starkInfo.cExpId))
    {
        h_deviceArgs.expType = 0;
    }
    else if (dests[0].params[0].parserParams.expId == int64_t(setupCtx.starkInfo.friExpId))
    {
        h_deviceArgs.expType = 1;
    }
    else
    {

        h_deviceArgs.expType = 2;
    }

    h_deviceArgs.boundSize = setupCtx.starkInfo.boundaries.size();

    // Dests
    DestGPU *dests_aux = new DestGPU[dests.size()];
    h_deviceArgs.nDests = dests.size();
    for (uint64_t i = 0; i < dests.size(); ++i)
    {
        dests_aux[i].dest_gpu = dests[i].dest_gpu;
        dests_aux[i].offset = dests[i].offset;
        dests_aux[i].dim = dests[i].dim;
        dests_aux[i].nParams = dests[i].params.size();
        dests_aux[i].domainSize = dests[i].domainSize;
        assert(dests_aux[i].nParams <= nParamsMax);
        dests_aux[i].params = new ParamsGPU[dests[i].params.size()]; // rick

        for (uint64_t j = 0; j < dests_aux[i].nParams; ++j)
        {
            dests_aux[i].params[j].dim = dests[i].params[j].dim;
            dests_aux[i].params[j].stage = dests[i].params[j].stage;
            dests_aux[i].params[j].stagePos = dests[i].params[j].stagePos;
            dests_aux[i].params[j].polsMapId = dests[i].params[j].polsMapId;
            dests_aux[i].params[j].rowOffsetIndex = dests[i].params[j].rowOffsetIndex;
            dests_aux[i].params[j].inverse = dests[i].params[j].inverse;
            dests_aux[i].params[j].batch = dests[i].params[j].batch;
            dests_aux[i].params[j].op = dests[i].params[j].op;
            dests_aux[i].params[j].value = dests[i].params[j].value;
            dests_aux[i].params[j].parserParams.stage = dests[i].params[j].parserParams.stage;
            dests_aux[i].params[j].parserParams.expId = dests[i].params[j].parserParams.expId;
            dests_aux[i].params[j].parserParams.nTemp1 = dests[i].params[j].parserParams.nTemp1;
            assert(dests_aux[i].params[j].parserParams.nTemp1 < nTemp1Max);
            dests_aux[i].params[j].parserParams.nTemp3 = dests[i].params[j].parserParams.nTemp3;
            assert(dests_aux[i].params[j].parserParams.nTemp3 < nTemp3Max);
            dests_aux[i].params[j].parserParams.nOps = dests[i].params[j].parserParams.nOps;
            dests_aux[i].params[j].parserParams.opsOffset = dests[i].params[j].parserParams.opsOffset;
            dests_aux[i].params[j].parserParams.nArgs = dests[i].params[j].parserParams.nArgs;
            dests_aux[i].params[j].parserParams.argsOffset = dests[i].params[j].parserParams.argsOffset;
            dests_aux[i].params[j].parserParams.constPolsOffset = dests[i].params[j].parserParams.constPolsOffset;
            dests_aux[i].params[j].parserParams.cmPolsOffset = dests[i].params[j].parserParams.cmPolsOffset;
            dests_aux[i].params[j].parserParams.challengesOffset = dests[i].params[j].parserParams.challengesOffset;
            dests_aux[i].params[j].parserParams.publicsOffset = dests[i].params[j].parserParams.publicsOffset;
            dests_aux[i].params[j].parserParams.airgroupValuesOffset = dests[i].params[j].parserParams.airgroupValuesOffset;
            dests_aux[i].params[j].parserParams.airValuesOffset = dests[i].params[j].parserParams.airValuesOffset;
            dests_aux[i].params[j].parserParams.firstRow = dests[i].params[j].parserParams.firstRow;
            dests_aux[i].params[j].parserParams.lastRow = dests[i].params[j].parserParams.lastRow;
            dests_aux[i].params[j].parserParams.destDim = dests[i].params[j].parserParams.destDim;
            dests_aux[i].params[j].parserParams.destId = dests[i].params[j].parserParams.destId;
            dests_aux[i].params[j].parserParams.imPol = dests[i].params[j].parserParams.imPol;
        }
    }

    // Dests
    DestGPU *d_dests = new DestGPU[h_deviceArgs.nDests];
    for (int i = 0; i < h_deviceArgs.nDests; ++i)
    {
        d_dests[i].dest_gpu = dests_aux[i].dest_gpu;
        d_dests[i].offset = dests_aux[i].offset;
        d_dests[i].dim = dests_aux[i].dim;
        d_dests[i].nParams = dests_aux[i].nParams;
        d_dests[i].domainSize = dests_aux[i].domainSize;
        hipMalloc(&d_dests[i].params, d_dests[i].nParams * sizeof(ParamsGPU));
        if (d_dests[i].nParams > 0)
            dest_params.push_back(d_dests[i].params);
        hipMemcpy(d_dests[i].params, dests_aux[i].params, d_dests[i].nParams * sizeof(ParamsGPU), hipMemcpyHostToDevice);
        CHECKCUDAERR(hipGetLastError());


    }
    for (int i = 0; i < dests.size(); i++)
    {
        delete[] dests_aux[i].params;
    }
    delete[] dests_aux;
    DestGPU *d_dests_;
    hipMalloc(&d_dests_, h_deviceArgs.nDests * sizeof(DestGPU));
    hipMemcpy(d_dests_, d_dests, h_deviceArgs.nDests * sizeof(DestGPU), hipMemcpyHostToDevice);
    CHECKCUDAERR(hipGetLastError());

    delete[] d_dests;
    h_deviceArgs.dests = d_dests_;

    // non polnomial arguments
    h_deviceArgs.nChallenges = setupCtx.starkInfo.challengesMap.size();
    h_deviceArgs.nNumbers = parserArgs.nNumbers;
    h_deviceArgs.nPublics = setupCtx.starkInfo.nPublics;
    h_deviceArgs.nEvals = setupCtx.starkInfo.evMap.size();
    h_deviceArgs.nAirgroupValues = setupCtx.starkInfo.airgroupValuesMap.size();
    h_deviceArgs.nAirValues = setupCtx.starkInfo.airValuesMap.size();
    h_deviceArgs.nProofValues = setupCtx.starkInfo.proofValuesMap.size();
    // Expressions bin
    h_deviceArgs.nOpsTotal = parserArgs.nOpsTotal;
    h_deviceArgs.nArgsTotal = parserArgs.nArgsTotal;

    // bufferT_
    h_deviceArgs.nBlocks = nBlocks;

    hipMemcpy(h_deviceArgs.cmPolsInfo, h_deviceArgs.cmPolsInfo, 3 * h_deviceArgs.cmPolsInfoSize * sizeof(uint64_t), hipMemcpyHostToDevice);
    CHECKCUDAERR(hipGetLastError());

    hipMemcpy(h_deviceArgs.zi, setupCtx.proverHelpers.zi, h_deviceArgs.boundSize * h_deviceArgs.NExtended * sizeof(Goldilocks::Element), hipMemcpyHostToDevice); // cal copiar cada cop?
    CHECKCUDAERR(hipGetLastError());

    hipMemcpy(h_deviceArgs.x_n, setupCtx.proverHelpers.x_n, h_deviceArgs.N * sizeof(Goldilocks::Element), hipMemcpyHostToDevice);                                // cal cada cop? no es pot transportar?
    CHECKCUDAERR(hipGetLastError());

    hipMemcpy(h_deviceArgs.x_2ns, setupCtx.proverHelpers.x_2ns, h_deviceArgs.NExtended * sizeof(Goldilocks::Element), hipMemcpyHostToDevice);                    // cal cada cop? no es pot transportar?
    CHECKCUDAERR(hipGetLastError());

    hipMemcpy(h_deviceArgs.challenges, params.challenges, h_deviceArgs.nChallenges * FIELD_EXTENSION * sizeof(Goldilocks::Element), hipMemcpyHostToDevice);
    CHECKCUDAERR(hipGetLastError());

    hipMemcpy(h_deviceArgs.numbers, (Goldilocks::Element *)parserArgs.numbers, h_deviceArgs.nNumbers * sizeof(Goldilocks::Element), hipMemcpyHostToDevice);
    CHECKCUDAERR(hipGetLastError());

    hipMemcpy(h_deviceArgs.publics, params.publicInputs, h_deviceArgs.nPublics * sizeof(Goldilocks::Element), hipMemcpyHostToDevice);
    CHECKCUDAERR(hipGetLastError());

    hipMemcpy(h_deviceArgs.evals, params.evals, h_deviceArgs.nEvals * FIELD_EXTENSION * sizeof(Goldilocks::Element), hipMemcpyHostToDevice);
    CHECKCUDAERR(hipGetLastError());


    Goldilocks::Element *airgroupValues_aux = new Goldilocks::Element[h_deviceArgs.nAirgroupValues * FIELD_EXTENSION];
    uint64_t p = 0;
    for (uint64_t i = 0; i < setupCtx.starkInfo.airgroupValuesMap.size(); ++i)
    {

        if (setupCtx.starkInfo.airgroupValuesMap[i].stage == 1)
        {
            airgroupValues_aux[i * FIELD_EXTENSION] = params.airgroupValues[p];
            airgroupValues_aux[i * FIELD_EXTENSION + 1] = Goldilocks::zero();
            airgroupValues_aux[i * FIELD_EXTENSION + 2] = Goldilocks::zero();
        }
        else
        {
            airgroupValues_aux[i * FIELD_EXTENSION] = params.airgroupValues[p];
            airgroupValues_aux[i * FIELD_EXTENSION + 1] = params.airgroupValues[p + 1];
            airgroupValues_aux[i * FIELD_EXTENSION + 2] = params.airgroupValues[p + 2];
        }

        if (setupCtx.starkInfo.airgroupValuesMap[i].stage == 1)
        {
            p += 1;
        }
        else
        {
            p += 3;
        }
    }
    hipMemcpy(h_deviceArgs.airgroupValues, airgroupValues_aux, h_deviceArgs.nAirgroupValues * FIELD_EXTENSION * sizeof(Goldilocks::Element), hipMemcpyHostToDevice);
    CHECKCUDAERR(hipGetLastError());
    delete[] airgroupValues_aux;

    Goldilocks::Element *airValues_aux = new Goldilocks::Element[h_deviceArgs.nAirValues * FIELD_EXTENSION];
    p = 0;
    for (uint64_t i = 0; i < setupCtx.starkInfo.airValuesMap.size(); ++i)
    {

        if (setupCtx.starkInfo.airValuesMap[i].stage == 1)
        {
            airValues_aux[i * FIELD_EXTENSION] = params.airValues[p];
            airValues_aux[i * FIELD_EXTENSION + 1] = Goldilocks::zero();
            airValues_aux[i * FIELD_EXTENSION + 2] = Goldilocks::zero();
        }
        else
        {
            airValues_aux[i * FIELD_EXTENSION] = params.airValues[p];
            airValues_aux[i * FIELD_EXTENSION + 1] = params.airValues[p + 1];
            airValues_aux[i * FIELD_EXTENSION + 2] = params.airValues[p + 2];
        }

        if (setupCtx.starkInfo.airValuesMap[i].stage == 1)
        {
            p += 1;
        }
        else
        {
            p += 3;
        }
    }
    hipMemcpy(h_deviceArgs.airValues, airValues_aux, h_deviceArgs.nAirValues * FIELD_EXTENSION * sizeof(Goldilocks::Element), hipMemcpyHostToDevice);
    CHECKCUDAERR(hipGetLastError());
    delete[] airValues_aux;
    Goldilocks::Element *proofValues_aux = new Goldilocks::Element[h_deviceArgs.nProofValues * FIELD_EXTENSION];
    p = 0;
    for (uint64_t i = 0; i < setupCtx.starkInfo.proofValuesMap.size(); ++i)
    {

        if (setupCtx.starkInfo.proofValuesMap[i].stage == 1)
        {
            proofValues_aux[i * FIELD_EXTENSION] = params.proofValues[p];
            proofValues_aux[i * FIELD_EXTENSION + 1] = Goldilocks::zero();
            proofValues_aux[i * FIELD_EXTENSION + 2] = Goldilocks::zero();
        }
        else
        {
            proofValues_aux[i * FIELD_EXTENSION] = params.proofValues[p];
            proofValues_aux[i * FIELD_EXTENSION + 1] = params.proofValues[p + 1];
            proofValues_aux[i * FIELD_EXTENSION + 2] = params.proofValues[p + 2];
        }

        if (setupCtx.starkInfo.proofValuesMap[i].stage == 1)
        {
            p += 1;
        }
        else
        {
            p += 3;
        }
    }

    hipMemcpy(h_deviceArgs.proofValues, proofValues_aux, h_deviceArgs.nProofValues * FIELD_EXTENSION * sizeof(Goldilocks::Element), hipMemcpyHostToDevice);
    CHECKCUDAERR(hipGetLastError());
    delete[] proofValues_aux;
    hipMemcpy(h_deviceArgs.ops, parserArgs.ops, h_deviceArgs.nOpsTotal * sizeof(uint8_t), hipMemcpyHostToDevice);
    CHECKCUDAERR(hipGetLastError());
    hipMemcpy(h_deviceArgs.args, parserArgs.args, h_deviceArgs.nArgsTotal * sizeof(uint16_t), hipMemcpyHostToDevice);
    CHECKCUDAERR(hipGetLastError());

    h_deviceArgs.constPols = h_deviceArgs.domainExtended ? params_gpu.pConstPolsExtendedTreeAddress : params_gpu.pConstPolsAddress;
    h_deviceArgs.trace = params_gpu.trace;
    h_deviceArgs.aux_trace = params_gpu.aux_trace;
    h_deviceArgs.xDivXSub = params_gpu.xDivXSub;
    h_deviceArgs.customCommits = params_gpu.pCustomCommitsFixed;

    // Allocate memory for the struct on the device
    hipMalloc(&d_deviceArgs, sizeof(DeviceArguments));
    hipMemcpy(d_deviceArgs, &h_deviceArgs, sizeof(DeviceArguments), hipMemcpyHostToDevice);
    CHECKCUDAERR(hipGetLastError());
}

void ExpressionsGPU::calculateExpressions_gpu(StepsParams &params, StepsParams &params_gpu, ParserArgs &parserArgs, std::vector<Dest> dests, uint64_t domainSize)
{

    CHECKCUDAERR(hipDeviceSynchronize());
    double time = omp_get_wtime();
    setBufferTInfo(domainSize, params, params_gpu, parserArgs, dests);
    CHECKCUDAERR(hipGetLastError());
    CHECKCUDAERR(hipDeviceSynchronize());
    time = omp_get_wtime() - time;
    std::cout << "goal2_ setBufferTInfo time: " << time << std::endl;

    CHECKCUDAERR(hipDeviceSynchronize());
    time = omp_get_wtime();
    dim3 nBlocks = h_deviceArgs.nBlocks;
    dim3 nThreads = h_deviceArgs.nrowsPack;
    std::cout << "goal2_ nBlocks: " << nBlocks.x << std::endl;
    //computeExpressions_explore_<<<nBlocks, nThreads>>>(d_deviceArgs);
    computeExpressions_<<<nBlocks, nThreads>>>(d_deviceArgs);
    CHECKCUDAERR(hipGetLastError());
    CHECKCUDAERR(hipDeviceSynchronize());
    time = omp_get_wtime() - time;
    std::cout << "goal2_ de computeExpressions: " << time << std::endl;

    CHECKCUDAERR(hipDeviceSynchronize());
    time = omp_get_wtime();
    std::cout << " nDests: " << h_deviceArgs.nDests << std::endl;
    for (uint32_t i = 0; i < h_deviceArgs.nDests; ++i)
    {
        if (dests[i].dest != NULL)
        {
            std::cout << " copy size: " << domainSize * FIELD_EXTENSION * sizeof(Goldilocks::Element) << std::endl;
            std::cout << " domainSize: " << dests[i].domainSize << std::endl;    
            Goldilocks::Element *dest = new Goldilocks::Element[dests[i].domainSize * FIELD_EXTENSION];
            hipMemcpy(dest, dests[i].dest_gpu, dests[i].domainSize * FIELD_EXTENSION * sizeof(Goldilocks::Element), hipMemcpyDeviceToHost);
            memcpy(dests[i].dest, dest, dests[i].domainSize * FIELD_EXTENSION * sizeof(Goldilocks::Element));
        }
    }
    CHECKCUDAERR(hipGetLastError());
    CHECKCUDAERR(hipDeviceSynchronize());
    time = omp_get_wtime() - time;
    std::cout << "goal2_ de hipMemcpy dests time: " << time << std::endl;

    CHECKCUDAERR(hipDeviceSynchronize());
    time = omp_get_wtime();
    freeDeviceArguments();
    CHECKCUDAERR(hipGetLastError());
    CHECKCUDAERR(hipDeviceSynchronize());
    time = omp_get_wtime() - time;
    std::cout << "goal2_ freeDeviceArguments time: " << time << std::endl;
}

void ExpressionsGPU::freeDeviceArguments()
{
    for (std::vector<ParamsGPU *>::iterator it = dest_params.begin(); it != dest_params.end(); ++it)
    {
        hipFree(*it);
    }
    dest_params.clear();
    hipFree(h_deviceArgs.dests);
    hipFree(d_deviceArgs);
}

__device__ __noinline__ void storeOnePolynomial__(DeviceArguments *d_deviceArgs, gl64_t *destVals, uint64_t row, uint32_t idest)
{
    if (row+blockIdx.x >= d_deviceArgs->dests[idest].domainSize){ //rick
        return;
    }
    if (d_deviceArgs->dests[idest].dim == 1)
    {
        uint64_t offset = d_deviceArgs->dests[idest].offset != 0 ? d_deviceArgs->dests[idest].offset : 1;
        gl64_t::copy_gpu((gl64_t *)&d_deviceArgs->dests[idest].dest_gpu[row * offset], uint64_t(offset), &destVals[0], false);
    }
    else
    {
        uint64_t offset = d_deviceArgs->dests[idest].offset != 0 ? d_deviceArgs->dests[idest].offset : FIELD_EXTENSION;
        gl64_t::copy_gpu((gl64_t *)&d_deviceArgs->dests[idest].dest_gpu[row * offset], uint64_t(offset), &destVals[0], false);
        gl64_t::copy_gpu((gl64_t *)&d_deviceArgs->dests[idest].dest_gpu[row * offset + 1], uint64_t(offset), &destVals[d_deviceArgs->nrowsPack], false);
        gl64_t::copy_gpu((gl64_t *)&d_deviceArgs->dests[idest].dest_gpu[row * offset + 2], uint64_t(offset), &destVals[2 * d_deviceArgs->nrowsPack], false);
    }
}

__device__ __noinline__ void copyPolynomial__(DeviceArguments *d_deviceArgs, gl64_t *destVals, bool inverse, uint64_t dim, gl64_t *temp)
{
    int idx = threadIdx.x;
    if (dim == 1)
    {
        if (inverse)
        {
            destVals[idx] = temp[idx].reciprocal();
        }
        else
        {
            destVals[idx] = temp[idx];
        }
    }
    else if (dim == FIELD_EXTENSION)
    {
        if (inverse)
        {
            Goldilocks3GPU::Element aux;
            aux[0] = temp[idx];
            aux[1] = temp[d_deviceArgs->nrowsPack + idx];
            aux[2] = temp[2 * d_deviceArgs->nrowsPack + idx];
            Goldilocks3GPU::inv(aux, aux);
            destVals[idx] = aux[0];
            destVals[d_deviceArgs->nrowsPack + idx] = aux[1];
            destVals[2 * d_deviceArgs->nrowsPack + idx] = aux[2];
        }
        else
        {
            destVals[idx] = temp[idx];
            destVals[d_deviceArgs->nrowsPack + idx] = temp[d_deviceArgs->nrowsPack + idx];
            destVals[2 * d_deviceArgs->nrowsPack + idx] = temp[2 * d_deviceArgs->nrowsPack + idx];
        }
    }
}

__device__ __noinline__ void loadPolynomials__(DeviceArguments *d_deviceArgs, uint64_t row, uint32_t iBlock)
{

    bool domainExtended = d_deviceArgs->domainExtended;
    uint64_t *nextStrides = d_deviceArgs->nextStrides;
    uint64_t *nColsStages = d_deviceArgs->nColsStages;
    uint64_t *nColsStagesAcc = d_deviceArgs->nColsStagesAcc;
    Goldilocks::Element *d_bufferT_ = &d_deviceArgs->bufferT_[iBlock * d_deviceArgs->bufferSize];

#pragma unroll 1
    for (uint64_t k = 0; k < d_deviceArgs->constPolsSize; ++k)
    {
        Goldilocks::Element *constPols = domainExtended ? &d_deviceArgs->constPols[2] : d_deviceArgs->constPols;
        for (uint64_t o = 0; o < d_deviceArgs->nOpenings; ++o)
        {
            uint64_t l = (row + threadIdx.x + nextStrides[o]) % d_deviceArgs->domainSize;
            d_bufferT_[(nColsStagesAcc[d_deviceArgs->ns * o] + k) * d_deviceArgs->nrowsPack + threadIdx.x] = constPols[l * nColsStages[0] + k];
        }
    }

#pragma unroll 1
    for (uint64_t k = 0; k < d_deviceArgs->cmPolsInfoSize; ++k)
    {
        uint64_t *cmPolsInfo = d_deviceArgs->cmPolsInfo;
        uint64_t stage = cmPolsInfo[k * 3];
        uint64_t stagePos = cmPolsInfo[k * 3 + 1];
        for (uint64_t d = 0; d < cmPolsInfo[k * 3 + 2]; ++d)
        {
            for (uint64_t o = 0; o < d_deviceArgs->nOpenings; ++o)
            {
                uint64_t l = (row + threadIdx.x + nextStrides[o]) % d_deviceArgs->domainSize;
                if (stage == 1 && !d_deviceArgs->domainExtended)
                {
                    d_bufferT_[(nColsStagesAcc[d_deviceArgs->ns * o + stage] + (stagePos + d)) * d_deviceArgs->nrowsPack + threadIdx.x] = d_deviceArgs->trace[l * nColsStages[stage] + stagePos + d];
                }
                else
                {
                    uint64_t *offsetsStages = d_deviceArgs->offsetsStages;
                    d_bufferT_[(nColsStagesAcc[d_deviceArgs->ns * o + stage] + (stagePos + d)) * d_deviceArgs->nrowsPack + threadIdx.x] = d_deviceArgs->aux_trace[offsetsStages[stage] + l * nColsStages[stage] + stagePos + d];
                }
            }
        }
    }

#pragma unroll 1
    for (uint64_t k = 0; k < d_deviceArgs->customCommitsCols; ++k)
    {
        uint64_t stage = d_deviceArgs->ns - 1; // rick: harcoded one single custom commit
        for (uint64_t o = 0; o < d_deviceArgs->nOpenings; ++o)
        {
            uint64_t l = (row + threadIdx.x + nextStrides[o]) % d_deviceArgs->domainSize;
            uint64_t offset = 0;
            if (d_deviceArgs->domainExtended)
            {
                offset = d_deviceArgs->N * d_deviceArgs->customCommitsCols;
            }
            d_bufferT_[(nColsStagesAcc[d_deviceArgs->ns * o + stage] + k) * d_deviceArgs->nrowsPack + threadIdx.x] = d_deviceArgs->customCommits[offset + l * nColsStages[stage] + k];
        }
    }

    if (d_deviceArgs->expType == 0)
    {
#pragma unroll 1
        for (uint64_t d = 0; d < d_deviceArgs->boundSize; ++d)
        {
            d_bufferT_[(nColsStagesAcc[d_deviceArgs->ns * d_deviceArgs->nOpenings] + d + 1) * d_deviceArgs->nrowsPack + threadIdx.x] = d_deviceArgs->zi[row + threadIdx.x + d * d_deviceArgs->domainSize];
        }
        d_bufferT_[(nColsStagesAcc[d_deviceArgs->ns * d_deviceArgs->nOpenings]) * d_deviceArgs->nrowsPack + threadIdx.x] = d_deviceArgs->x_2ns[row + threadIdx.x];
    }
    else if (d_deviceArgs->expType == 1)
    {
#pragma unroll 1
        for (uint64_t d = 0; d < d_deviceArgs->nOpenings; ++d)
        {
            for (uint64_t k = 0; k < FIELD_EXTENSION; ++k)
            {
                d_bufferT_[(nColsStagesAcc[d_deviceArgs->ns * d_deviceArgs->nOpenings] + d * FIELD_EXTENSION + k) * d_deviceArgs->nrowsPack + threadIdx.x] = d_deviceArgs->xDivXSub[(row + threadIdx.x + d * d_deviceArgs->domainSize) * FIELD_EXTENSION + k];
            }
        }
    }
    else
    {
        d_bufferT_[(nColsStagesAcc[d_deviceArgs->ns * d_deviceArgs->nOpenings]) * d_deviceArgs->nrowsPack + threadIdx.x] = d_deviceArgs->x_n[row + threadIdx.x];
    }
}

__device__ __noinline__ void multiplyPolynomials__(DeviceArguments *deviceArgs, DestGPU &dest, gl64_t *destVals)
{
    if (dest.dim == 1)
    {
        gl64_t::op_gpu(2, &destVals[0], &destVals[0], false, &destVals[FIELD_EXTENSION * deviceArgs->nrowsPack], false);
    }
    else
    {
        assert(blockDim.x <= 128);
        __shared__ gl64_t vals[FIELD_EXTENSION * 128]; // rick: corregir
        if (dest.params[0].dim == FIELD_EXTENSION && dest.params[1].dim == FIELD_EXTENSION)
        {
            Goldilocks3GPU::op_gpu(2, &vals[0], &destVals[0], false, &destVals[FIELD_EXTENSION * deviceArgs->nrowsPack], false);
        }
        else if (dest.params[0].dim == FIELD_EXTENSION && dest.params[1].dim == 1)
        {
            Goldilocks3GPU::op_31_gpu(2, &vals[0], &destVals[0], false, &destVals[FIELD_EXTENSION * deviceArgs->nrowsPack], false);
        }
        else
        {
            Goldilocks3GPU::op_31_gpu(2, &vals[0], &destVals[FIELD_EXTENSION * deviceArgs->nrowsPack], false, &destVals[0], false);
        }
        gl64_t::copy_gpu(&destVals[0], &vals[0], false);
        gl64_t::copy_gpu(&destVals[deviceArgs->nrowsPack], &vals[deviceArgs->nrowsPack], false);
        gl64_t::copy_gpu(&destVals[2 * deviceArgs->nrowsPack], &vals[2 * deviceArgs->nrowsPack], false);
    }
}

__global__ __launch_bounds__(128) void computeExpressions_explore_(DeviceArguments *d_deviceArgs)
{

    int chunk_idx = blockIdx.x;
    assert(d_deviceArgs->nrowsPack == blockDim.x);
    uint64_t nchunks = d_deviceArgs->domainSize / blockDim.x;


    __shared__ gl64_t *expressions_params[10];

    if (threadIdx.x == 0)
    {
        expressions_params[0] = (gl64_t *)(&d_deviceArgs->bufferT_[blockIdx.x * d_deviceArgs->bufferSize]);
        expressions_params[1] = (gl64_t *)(&d_deviceArgs->tmp1[blockIdx.x * d_deviceArgs->tmp1Size]);
        expressions_params[2] = (gl64_t *)d_deviceArgs->publics;
        expressions_params[3] = (gl64_t *)d_deviceArgs->numbers;
        expressions_params[4] = (gl64_t *)d_deviceArgs->airValues;
        expressions_params[5] = (gl64_t *)d_deviceArgs->proofValues;
        expressions_params[6] = (gl64_t *)(&d_deviceArgs->tmp3[blockIdx.x * d_deviceArgs->tmp3Size]);
        expressions_params[7] = (gl64_t *)d_deviceArgs->airgroupValues;
        expressions_params[8] = (gl64_t *)d_deviceArgs->challenges;
        expressions_params[9] = (gl64_t *)d_deviceArgs->evals;
    }
    __syncthreads();

    while (chunk_idx < nchunks)
    {
        uint64_t i = chunk_idx * blockDim.x;
        bool print = false ; //threadIdx.x == 0 && blockIdx.x == 0 && i==0;
        loadPolynomials__(d_deviceArgs, i, blockIdx.x);

        if(print) printf(" Ndests: %d\n", d_deviceArgs->nDests);

#pragma unroll 1
        for (uint64_t j = 0; j < d_deviceArgs->nDests; ++j)
        {
            if(print) printf(" dest: %lu\n", j);

            for (uint64_t k = 0; k < d_deviceArgs->dests[j].nParams; ++k)
            {
                if(print) printf(" param: %lu of %d\n", k, d_deviceArgs->dests[j].nParams);

                gl64_t *destVals = (gl64_t *)(&d_deviceArgs->destVals[blockIdx.x * d_deviceArgs->destValsSize]);
                uint64_t *nColsStagesAcc = d_deviceArgs->nColsStagesAcc;


                if(print) printf(" nOps: %d\n", d_deviceArgs->dests[j].params[k].parserParams.nOps);
                if(print) printf(" offset: %d\n", d_deviceArgs->dests[j].params[k].parserParams.opsOffset);
                if(print) printf(" nArgs: %d\n", d_deviceArgs->dests[j].params[k].parserParams.nArgs);
                if(print) printf(" argsOffset: %d\n", d_deviceArgs->dests[j].params[k].parserParams.argsOffset);
                if(print) printf(" nOpsTotal: %d\n", d_deviceArgs->nOpsTotal);
                for (uint64_t kk = 0; kk < d_deviceArgs->dests[j].params[k].parserParams.nOps; ++kk)
                {
                    assert(d_deviceArgs->dests[j].params[k].parserParams.opsOffset + kk < d_deviceArgs->nOpsTotal);
                }
            }
        }       
        chunk_idx += gridDim.x;
    }
}
__global__ __launch_bounds__(128) void computeExpressions_(DeviceArguments *d_deviceArgs)
{

    int chunk_idx = blockIdx.x;
    assert(d_deviceArgs->nrowsPack == blockDim.x);
    uint64_t nchunks = d_deviceArgs->domainSize / blockDim.x;


    __shared__ gl64_t *expressions_params[10];

    if (threadIdx.x == 0)
    {
        expressions_params[0] = (gl64_t *)(&d_deviceArgs->bufferT_[blockIdx.x * d_deviceArgs->bufferSize]);
        expressions_params[1] = (gl64_t *)(&d_deviceArgs->tmp1[blockIdx.x * d_deviceArgs->tmp1Size]);
        expressions_params[2] = (gl64_t *)d_deviceArgs->publics;
        expressions_params[3] = (gl64_t *)d_deviceArgs->numbers;
        expressions_params[4] = (gl64_t *)d_deviceArgs->airValues;
        expressions_params[5] = (gl64_t *)d_deviceArgs->proofValues;
        expressions_params[6] = (gl64_t *)(&d_deviceArgs->tmp3[blockIdx.x * d_deviceArgs->tmp3Size]);
        expressions_params[7] = (gl64_t *)d_deviceArgs->airgroupValues;
        expressions_params[8] = (gl64_t *)d_deviceArgs->challenges;
        expressions_params[9] = (gl64_t *)d_deviceArgs->evals;
    }
    __syncthreads();

    while (chunk_idx < nchunks)
    {
        uint64_t i = chunk_idx * blockDim.x;
        bool print = false; //threadIdx.x == 0 && blockIdx.x == 0 && i==0;
        loadPolynomials__(d_deviceArgs, i, blockIdx.x);

        if(print) printf(" Ndests: %d\n", d_deviceArgs->nDests);

#pragma unroll 1
        for (uint64_t j = 0; j < d_deviceArgs->nDests; ++j)
        {
            if(print) printf(" dest: %lu\n", j);

            for (uint64_t k = 0; k < d_deviceArgs->dests[j].nParams; ++k)
            {
                if(print) printf(" param: %lu of %d\n", k, d_deviceArgs->dests[j].nParams);

                gl64_t *destVals = (gl64_t *)(&d_deviceArgs->destVals[blockIdx.x * d_deviceArgs->destValsSize]);
                uint64_t *nColsStagesAcc = d_deviceArgs->nColsStagesAcc;

                if (d_deviceArgs->dests[j].params[k].op == opType::cm || d_deviceArgs->dests[j].params[k].op == opType::const_)
                {
                    uint64_t openingPointIndex = d_deviceArgs->dests[j].params[k].rowOffsetIndex;
                    uint64_t buffPos = d_deviceArgs->ns * openingPointIndex + d_deviceArgs->dests[j].params[k].stage;
                    uint64_t stagePos = d_deviceArgs->dests[j].params[k].stagePos;
                    copyPolynomial__(d_deviceArgs, &destVals[k * FIELD_EXTENSION * blockDim.x], d_deviceArgs->dests[j].params[k].inverse, d_deviceArgs->dests[j].params[k].dim, &expressions_params[0][(nColsStagesAcc[buffPos] + stagePos) * blockDim.x]);
                    continue;
                }
                else if (d_deviceArgs->dests[j].params[k].op == opType::number)
                {
                    gl64_t val(d_deviceArgs->dests[j].params[k].value);
                    if (d_deviceArgs->dests[j].params[k].inverse)
                        val = val.reciprocal();
                    destVals[k * FIELD_EXTENSION * blockDim.x + threadIdx.x] = val;
                    continue;
                }

                if(print) printf(" nOps: %d\n", d_deviceArgs->dests[j].params[k].parserParams.nOps);
                if(print) printf(" offset: %d\n", d_deviceArgs->dests[j].params[k].parserParams.opsOffset);
                uint8_t *ops = &d_deviceArgs->ops[d_deviceArgs->dests[j].params[k].parserParams.opsOffset];
                uint16_t *args = &d_deviceArgs->args[d_deviceArgs->dests[j].params[k].parserParams.argsOffset];

                uint64_t i_args = 0;
                for (uint64_t kk = 0; kk < d_deviceArgs->dests[j].params[k].parserParams.nOps; ++kk)
                {
                    switch (ops[kk])
                    {
                    case 0:
                    {
                        // COPY dim1 to dim1
                        if (print)
                        {
                            // printf("Abans args: %d %d %d %d %d %d %d %d %d %d %d %d\n", args[i_args], args[i_args + 1], args[i_args + 2], args[i_args + 3], args[i_args + 4], args[i_args + 5], args[i_args + 6], args[i_args + 7], args[i_args + 8], args[i_args + 9], args[i_args + 10], args[i_args + 11]);
                        }
                        gl64_t::copy_gpu(&expressions_params[args[i_args]][(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * blockDim.x], &expressions_params[args[i_args + 3]][(nColsStagesAcc[args[i_args + 4]] + args[i_args + 5]) * (1 - args[i_args + 6]) * blockDim.x + args[i_args + 6] * args[i_args + 5]], args[i_args + 6]);
                        if (print)
                        {
                            // result
                            printf("Case 0\n");
                            printf("Op %lu of %d\n", kk, d_deviceArgs->dests[j].params[k].parserParams.nOps);
                            // printf("Arguments %lu\n", expressions_params[args[i_args + 3]][(nColsStagesAcc[args[i_args + 4]] + args[i_args + 5]) * d_deviceArgs->nrowsPack].get_val());
                            printf("Result: %lu\n", expressions_params[args[i_args]][(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * d_deviceArgs->nrowsPack].get_val());
                            // printf("args %d %d %d %d %d %d\n", args[i_args], args[i_args + 1], args[i_args + 2], args[i_args + 3], args[i_args + 4], args[i_args + 5]);
                        }

                        i_args += 7;
                        break;
                    }
                    case 1:
                    {
                        // OPERATION WITH DEST: dim1 - SRC0: dim1 - SRC1: dim1
                        gl64_t::op_gpu(args[i_args], &expressions_params[args[i_args + 1]][(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * blockDim.x], &expressions_params[args[i_args + 4]][(nColsStagesAcc[args[i_args + 5]] + args[i_args + 6]) * (1 - args[i_args + 7]) * blockDim.x + args[i_args + 7] * args[i_args + 6]], args[i_args + 7], &expressions_params[args[i_args + 8]][(nColsStagesAcc[args[i_args + 9]] + args[i_args + 10]) * (1 - args[i_args + 11]) * blockDim.x + args[i_args + 11] * args[i_args + 10]], args[i_args + 11]);
                        if (print)
                        {
                            // result
                            printf("Case 1\n");
                            printf("Op %lu of %d\n", kk, d_deviceArgs->dests[j].params[k].parserParams.nOps);
                            // printf("Buffer: %d %d %d \n", args[i_args + 1], args[i_args + 4], args[i_args + 8]);
                            printf("Result: %lu\n", expressions_params[args[i_args + 1]][(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * d_deviceArgs->nrowsPack].get_val());
                            // printf("args: %d %d %d %d %d %d %d %d %d %d %d %d\n", args[i_args], args[i_args + 1], args[i_args + 2], args[i_args + 3], args[i_args + 4], args[i_args + 5], args[i_args + 6], args[i_args + 7], args[i_args + 8], args[i_args + 9], args[i_args + 10], args[i_args + 11]);
                        }
                        i_args += 12;
                        break;
                    }
                    case 2:
                    {
                        // OPERATION WITH DEST: dim3 - SRC0: dim3 - SRC1: dim1
                        /*if( print){
                            printf("Abans args: %d %d %d %d %d %d %d %d %d %d %d %d\n", args[i_args], args[i_args + 1], args[i_args + 2], args[i_args + 3], args[i_args + 4], args[i_args + 5], args[i_args + 6], args[i_args + 7], args[i_args + 8], args[i_args + 9], args[i_args + 10], args[i_args + 11]);
                        }*/
                        Goldilocks3GPU::op_31_gpu(args[i_args], &expressions_params[args[i_args + 1]][(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * blockDim.x], &expressions_params[args[i_args + 4]][(nColsStagesAcc[args[i_args + 5]] + args[i_args + 6]) * (1 - args[i_args + 7]) * blockDim.x + args[i_args + 7] * args[i_args + 6]], args[i_args + 7], &expressions_params[args[i_args + 8]][(nColsStagesAcc[args[i_args + 9]] + args[i_args + 10]) * (1 - args[i_args + 11]) * blockDim.x + args[i_args + 11] * args[i_args + 10]], args[i_args + 11]);
                        if (print)
                        {
                            // result
                            printf("Case 2\n");
                            printf("Op %lu of %d\n", kk, d_deviceArgs->dests[j].params[k].parserParams.nOps);
                            // printf("Buffer: %d %d %d \n", args[i_args + 1], args[i_args + 4], args[i_args + 8]);
                            printf("Result: %lu %lu %lu\n",
                                   expressions_params[args[i_args + 1]][(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * d_deviceArgs->nrowsPack].get_val(),
                                   expressions_params[args[i_args + 1]][(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * d_deviceArgs->nrowsPack + 1].get_val(),
                                   expressions_params[args[i_args + 1]][(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * d_deviceArgs->nrowsPack + 2].get_val());
                        }
                        i_args += 12;
                        break;
                    }
                    case 3:
                    {
                        /*if( print){
                            printf("Abans args: %d %d %d %d %d %d %d %d %d %d %d %d\n", args[i_args], args[i_args + 1], args[i_args + 2], args[i_args + 3], args[i_args + 4], args[i_args + 5], args[i_args + 6], args[i_args + 7], args[i_args + 8], args[i_args + 9], args[i_args + 10], args[i_args + 11]);
                        }*/
                        // OPERATION WITH DEST: dim3 - SRC0: dim3 - SRC1: dim3
                        Goldilocks3GPU::op_gpu(args[i_args], &expressions_params[args[i_args + 1]][(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * blockDim.x], &expressions_params[args[i_args + 4]][(nColsStagesAcc[args[i_args + 5]] + args[i_args + 6]) * (1 - args[i_args + 7]) * blockDim.x + args[i_args + 7] * args[i_args + 6]], args[i_args + 7], &expressions_params[args[i_args + 8]][(nColsStagesAcc[args[i_args + 9]] + args[i_args + 10]) * (1 - args[i_args + 11]) * blockDim.x + args[i_args + 11] * args[i_args + 10]], args[i_args + 11]);
                        if (print)
                        {
                            // result
                            printf("Case 3\n");
                            printf("Op %lu of %d\n", kk, d_deviceArgs->dests[j].params[k].parserParams.nOps);
                            printf("Result: %lu %lu %lu\n",
                                   expressions_params[args[i_args + 1]][(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * d_deviceArgs->nrowsPack].get_val(),
                                   expressions_params[args[i_args + 1]][(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * d_deviceArgs->nrowsPack + 1].get_val(),
                                   expressions_params[args[i_args + 1]][(nColsStagesAcc[args[i_args + 2]] + args[i_args + 3]) * d_deviceArgs->nrowsPack + 2].get_val());
                        }
                        i_args += 12;
                        break;
                    }
                    case 4:
                    {
                        /*if( print){
                            printf("Abans args: %d %d %d %d %d %d %d %d %d %d %d %d\n", args[i_args], args[i_args + 1], args[i_args + 2], args[i_args + 3], args[i_args + 4], args[i_args + 5], args[i_args + 6], args[i_args + 7], args[i_args + 8], args[i_args + 9], args[i_args + 10], args[i_args + 11]);
                        }*/
                        // COPY dim3 to dim3
                        Goldilocks3GPU::copy_gpu(&expressions_params[args[i_args]][(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * blockDim.x], &expressions_params[args[i_args + 3]][(nColsStagesAcc[args[i_args + 4]] + args[i_args + 5]) * (1 - args[i_args + 6]) * blockDim.x + args[i_args + 6] * args[i_args + 5]], args[i_args + 6]);
                        if (print)
                        {
                            // result
                            printf("Case 4\n");
                            printf("Op %lu of %d\n", kk, d_deviceArgs->dests[j].params[k].parserParams.nOps);
                            printf("Result: %lu %lu %lu\n",
                                   expressions_params[args[i_args]][(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * d_deviceArgs->nrowsPack].get_val(),
                                   expressions_params[args[i_args]][(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * d_deviceArgs->nrowsPack + 1].get_val(),
                                   expressions_params[args[i_args]][(nColsStagesAcc[args[i_args + 1]] + args[i_args + 2]) * d_deviceArgs->nrowsPack + 2].get_val());
                        }
                        i_args += 7;
                        break;
                    }
                    default:
                    {
                        printf(" Wrong operation!\n");
                        assert(0);
                    }
                    }
                }

                if (d_deviceArgs->dests[j].params[k].parserParams.destDim == 1)
                {
                    copyPolynomial__(d_deviceArgs, &destVals[k * FIELD_EXTENSION * blockDim.x], d_deviceArgs->dests[j].params[k].inverse, d_deviceArgs->dests[j].params[k].parserParams.destDim, &expressions_params[1][d_deviceArgs->dests[j].params[k].parserParams.destId * blockDim.x]);
                }
                else
                {
                    copyPolynomial__(d_deviceArgs, &destVals[k * FIELD_EXTENSION * blockDim.x], d_deviceArgs->dests[j].params[k].inverse, d_deviceArgs->dests[j].params[k].parserParams.destDim, &expressions_params[6][d_deviceArgs->dests[j].params[k].parserParams.destId * FIELD_EXTENSION * blockDim.x]);
                }
            }

            gl64_t *destVals = (gl64_t *)(&d_deviceArgs->destVals[blockIdx.x * d_deviceArgs->destValsSize]);
            if (d_deviceArgs->dests[j].nParams == 2)
            {
                multiplyPolynomials__(d_deviceArgs, d_deviceArgs->dests[j], destVals);
            }
            storeOnePolynomial__(d_deviceArgs, destVals, i, j);
        }
        chunk_idx += gridDim.x;
    }
}
